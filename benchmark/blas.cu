#include <hipblas.h>
#include <hip/hip_complex.h>
#include <cstdio>


static const char *cublasGetErrorString(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
        default:
            return "<unknown>";
    }
    return "<unknown>";
}

#define checkCudaErrors(stmt) {                                 \
    hipError_t err = stmt;                            \
    if (err != hipSuccess) {                          \
      fprintf(stderr, "%s in file %s, function %s, line %i: %04d %s\n", #stmt, __FILE__, __FUNCTION__, __LINE__, err, hipGetErrorString(err)); \
      exit(1); \
    }                                                  \
}

#define checkCuttErrors(stmt) {                                 \
    cuttResult err = stmt;                            \
    if (err != CUTT_SUCCESS) {                          \
      fprintf(stderr, "%s in file %s, function %s, line %i.\n", #stmt, __FILE__, __FUNCTION__, __LINE__); \
      exit(1); \
    }                                                  \
}

#define checkBlasErrors(stmt) { \
    hipblasStatus_t err = stmt; \
    if (err != HIPBLAS_STATUS_SUCCESS) {                          \
      fprintf(stderr, "%s in file %s, function %s, line %i: %04d %s\n", #stmt, __FILE__, __FUNCTION__, __LINE__, err, cublasGetErrorString(err)); \
      exit(1); \
    } \
}

int main() {
    int nq = N_QUBIT;
    hipDoubleComplex* arr;
    hipDoubleComplex* mat;
    hipDoubleComplex* result;
    checkCudaErrors(hipMalloc(&arr, sizeof(hipDoubleComplex) << nq));
    checkCudaErrors(hipMalloc(&mat, sizeof(hipDoubleComplex) * 1024 * 1024));
    checkCudaErrors(hipMalloc(&result, sizeof(hipDoubleComplex) << nq));
    hipblasHandle_t handle;
    checkBlasErrors(hipblasCreate(&handle));
    // checkBlasErrors(hipblasSetMathMode(handle, HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION));
    int numElements = 1 << nq;
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0), beta = make_hipDoubleComplex(0.0, 0.0);
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    for (int K = 2; K < 1024; K <<= 1) {
        printf("K = %d\n", K);
        for (int i = 0; i < 100; i++) {
            checkCudaErrors(hipEventRecord(start));
            
            checkBlasErrors(hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                K, numElements / K, K, // M, N, K
                &alpha, mat, K, // alpha, a, lda
                arr, K, // b, ldb
                &beta, result, K // beta, c, ldc
            ));

            float time;
            checkCudaErrors(hipEventRecord(stop));
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            printf("%.10f ", time);
        }
        printf("\n");
    }
    return 0;
}
