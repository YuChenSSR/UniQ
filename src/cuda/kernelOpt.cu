#include "hip/hip_runtime.h"
#include "cuda/kernel.h"
#include <cstdio>
#include <assert.h>
#include <map>
#include <omp.h>
#include "gate.h"
#include "executor.h"
#include "dbg.h"
using namespace std;

static __shared__ cuCpx shm[1<<LOCAL_QUBIT_SIZE];
static __shared__ idx_t blockBias;

__device__ __constant__ value_t recRoot2 = 0.70710678118654752440084436210485; // more elegant way?
__constant__ KernelGate deviceGates[MAX_GATE];

std::vector<int*> loIdx_device;
std::vector<int*> shiftAt_device;


__device__ __forceinline__ void XSingle(int loIdx, int hiIdx) {
    cuCpx v = shm[loIdx];
    shm[loIdx] = shm[hiIdx];
    shm[hiIdx] = v;
}

__device__ __forceinline__ void YSingle(int loIdx, int hiIdx) {
    cuCpx lo = shm[loIdx];
    cuCpx hi = shm[hiIdx];
    
    shm[loIdx] = make_hipComplex(hi.y, -hi.x);
    shm[hiIdx] = make_hipComplex(-lo.y, lo.x);
}

__device__ __forceinline__ void ZHi(int hiIdx) {
    cuCpx v = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(-v.x, -v.y);
}


__device__ __forceinline__ void RXSingle(int loIdx, int hiIdx, value_t alpha, value_t beta) {
    cuCpx lo = shm[loIdx];
    cuCpx hi = shm[hiIdx];
    shm[loIdx] = make_hipComplex(alpha * lo.x + beta * hi.y, alpha * lo.y - beta * hi.x);
    shm[hiIdx] = make_hipComplex(alpha * hi.x + beta * lo.y, alpha * hi.y - beta * lo.x);
}

__device__ __forceinline__ void RYSingle(int loIdx, int hiIdx, value_t alpha, value_t beta) {
    cuCpx lo = shm[loIdx];
    cuCpx hi = shm[hiIdx];
    shm[loIdx] = make_hipComplex(alpha * lo.x - beta * hi.x, alpha * lo.y - beta * hi.y);
    shm[hiIdx] = make_hipComplex(beta * lo.x + alpha * hi.x, beta * lo.y + alpha * hi.y);
}

__device__ __forceinline__ void RZSingle(int loIdx, int hiIdx, value_t alpha, value_t beta){
    cuCpx lo = shm[loIdx];
    cuCpx hi = shm[hiIdx];
    shm[loIdx] = make_hipComplex(alpha * lo.x + beta * lo.y, alpha * lo.y - beta * lo.x);
    shm[hiIdx] = make_hipComplex(alpha * hi.x - beta * hi.y, alpha * hi.y + beta * hi.x);
}

__device__ __forceinline__ void RZLo(int loIdx, value_t alpha, value_t beta) {
    cuCpx lo = shm[loIdx];
    shm[loIdx] = make_hipComplex(alpha * lo.x + beta * lo.y, alpha * lo.y - beta * lo.x);
}

__device__ __forceinline__ void RZHi(int hiIdx, value_t alpha, value_t beta){
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(alpha * hi.x - beta * hi.y, alpha * hi.y + beta * hi.x);
}

#define COMPLEX_MULTIPLY_REAL(v0, v1) (v0.x * v1.x - v0.y * v1.y)
#define COMPLEX_MULTIPLY_IMAG(v0, v1) (v0.x * v1.y + v0.y * v1.x)

__device__ __forceinline__ void U1Hi(int hiIdx, cuCpx p) {
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(COMPLEX_MULTIPLY_REAL(hi, p), COMPLEX_MULTIPLY_IMAG(hi, p));
}

__device__ __forceinline__ void USingle(int loIdx, int hiIdx, cuCpx v00, cuCpx v01, cuCpx v10, cuCpx v11) {
    cuCpx lo = shm[loIdx];
    cuCpx hi = shm[hiIdx];
    shm[loIdx] = make_hipComplex(COMPLEX_MULTIPLY_REAL(lo, v00) + COMPLEX_MULTIPLY_REAL(hi, v01),
                               COMPLEX_MULTIPLY_IMAG(lo, v00) + COMPLEX_MULTIPLY_IMAG(hi, v01));
    shm[hiIdx] = make_hipComplex(COMPLEX_MULTIPLY_REAL(lo, v10) + COMPLEX_MULTIPLY_REAL(hi, v11),
                               COMPLEX_MULTIPLY_IMAG(lo, v10) + COMPLEX_MULTIPLY_IMAG(hi, v11));
}

__device__ __forceinline__ void HSingle(int loIdx, int hiIdx) {
    cuCpx lo = shm[loIdx];
    cuCpx hi = shm[hiIdx];
    shm[loIdx] = make_hipComplex(recRoot2 * (lo.x + hi.x), recRoot2 * (lo.y + hi.y));
    shm[hiIdx] = make_hipComplex(recRoot2 * (lo.x - hi.x), recRoot2 * (lo.y - hi.y));
}

__device__ __forceinline__ void SHi(int hiIdx) {
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(-hi.y, hi.x);
}

__device__ __forceinline__ void SDGHi(int hiIdx) {
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(hi.y, -hi.x);
}

__device__ __forceinline__ void THi(int hiIdx) {
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(recRoot2 * (hi.x - hi.y), recRoot2 * (hi.x + hi.y));
}

__device__ __forceinline__ void TDGHi(int hiIdx) {
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(recRoot2 * (hi.x + hi.y), recRoot2 * (hi.x - hi.y));
}
__device__ __forceinline__ void GIISingle(int loIdx, int hiIdx) {
    cuCpx lo = shm[loIdx];
    shm[loIdx] = make_hipComplex(-lo.y, lo.x);
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(-hi.y, hi.x);
}

__device__ __forceinline__ void GII(int idx) {
    cuCpx v = shm[idx];
    shm[idx] = make_hipComplex(-v.y, v.x);
}

__device__ __forceinline__ void GZZSingle(int loIdx, int hiIdx) {
    cuCpx lo = shm[loIdx];
    shm[loIdx] = make_hipComplex(-lo.x, -lo.y);
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(-hi.x, -hi.y);
}

__device__ __forceinline__ void GZZ(int idx) { 
    cuCpx v = shm[idx];
    shm[idx] = make_hipComplex(-v.x, -v.y);
}

__device__ __forceinline__ void GCCSingle(int loIdx, int hiIdx, cuCpx p) {
    cuCpx lo = shm[loIdx];
    shm[loIdx] = make_hipComplex(COMPLEX_MULTIPLY_REAL(lo, p), COMPLEX_MULTIPLY_IMAG(lo, p));
    cuCpx hi = shm[hiIdx];
    shm[hiIdx] = make_hipComplex(COMPLEX_MULTIPLY_REAL(hi, p), COMPLEX_MULTIPLY_IMAG(hi, p));
}

__device__ __forceinline__ void GCC(int idx, cuCpx p) {
    cuCpx v = shm[idx];
    shm[idx] = make_hipComplex(COMPLEX_MULTIPLY_REAL(v, p), COMPLEX_MULTIPLY_IMAG(v, p));
}

#define FOLLOW_NEXT(TYPE) \
case GateType::TYPE: // no break

#define CASE_CONTROL(TYPE, OP) \
case GateType::TYPE: { \
    assert(lo < 1024); \
    assert(hi < 1024); \
    OP; \
    lo += add; hi += add; \
    assert(lo < 1024); \
    assert(hi < 1024); \
    OP; \
    break; \
}

#define CASE_SINGLE(TYPE, OP) \
case GateType::TYPE: { \
    for (int task = 0; task < 4; task++) { \
        OP; \
        lo += add[task]; hi += add[task]; \
    } \
    break;\
}

#define CASE_LO_HI(TYPE, OP_LO, OP_HI) \
case GateType::TYPE: { \
    int m = 1 << LOCAL_QUBIT_SIZE; \
    if (!isHighBlock){ \
        for (int j = threadIdx.x; j < m; j += blockSize) { \
            OP_LO; \
        } \
    } else { \
        for (int j = threadIdx.x; j < m; j += blockSize) { \
            OP_HI; \
        } \
    } \
    break; \
}

#define CASE_SKIPLO_HI(TYPE, OP_HI) \
case GateType::TYPE: { \
    if (!isHighBlock) continue; \
    int m = 1 << LOCAL_QUBIT_SIZE; \
    for (int j = threadIdx.x; j < m; j += blockSize) { \
        OP_HI; \
    } \
    break; \
}

#define LOHI_SAME(TYPE, OP) \
case GateType::TYPE: { \
    int m = 1 << LOCAL_QUBIT_SIZE; \
    for (int j = threadIdx.x; j < m; j += blockSize) { \
        OP; \
    } \
    break; \
}

#define ID_BREAK() \
case GateType::ID: { \
    break; \
}

template <unsigned int blockSize>
__device__ void doCompute(int numGates, int* loArr, int* shiftAt) {
    for (int i = 0; i < numGates; i++) {
        int controlQubit = deviceGates[i].controlQubit;
        int targetQubit = deviceGates[i].targetQubit;
        char controlIsGlobal = deviceGates[i].controlIsGlobal;
        char targetIsGlobal = deviceGates[i].targetIsGlobal;
        if (deviceGates[i].type == GateType::CCX) {
            int encodeQubit = deviceGates[i].encodeQubit;
            int control2IsGlobal = deviceGates[i].control2IsGlobal;
            if (!control2IsGlobal) {
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                assert(!controlIsGlobal && !targetIsGlobal);
                assert(deviceGates[i].type == GateType::CCX);
                int maskTarget = (1 << targetQubit) - 1;
                for (int j = threadIdx.x; j < m; j += blockSize) {
                    int lo = ((j >> targetQubit) << (targetQubit + 1)) | (j & maskTarget);
                    if (!(lo >> controlQubit & 1) || !(lo >> encodeQubit & 1))
                        continue;
                    int hi = lo | (1 << targetQubit);
                    lo ^= lo >> 3 & 7;
                    hi ^= hi >> 3 & 7;
                    XSingle(lo, hi);
                }
                continue;
            }
            if (control2IsGlobal == 1 && !((blockIdx.x >> encodeQubit) & 1)) {
                continue;
            }
        }
        if (!controlIsGlobal) {
            if (!targetIsGlobal) {
                int lo = loArr[(controlQubit * 10 + targetQubit) << THREAD_DEP | threadIdx.x];
                int hi = lo ^ (1 << targetQubit) ^ (((1 << targetQubit) >> 3) & 7);
                int add = 512;
                if (controlQubit == 9 || targetQubit == 9) {
                    add = 256;
                    if (controlQubit == 8 || targetQubit == 8)
                        add = 128;
                }
                switch (deviceGates[i].type) {
                    FOLLOW_NEXT(CCX)
                    CASE_CONTROL(CNOT, XSingle(lo, hi))
                    CASE_CONTROL(CY, YSingle(lo, hi))
                    CASE_CONTROL(CZ, ZHi(hi))
                    CASE_CONTROL(CRX, RXSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i01))
                    CASE_CONTROL(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                    CASE_CONTROL(CU1, U1Hi(hi, make_hipComplex(deviceGates[i].r11, deviceGates[i].i11)))
                    CASE_CONTROL(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                    CASE_CONTROL(CU, USingle(lo, hi, make_hipComplex(deviceGates[i].r00, deviceGates[i].i00), make_hipComplex(deviceGates[i].r01, deviceGates[i].i01), make_hipComplex(deviceGates[i].r10, deviceGates[i].i10), make_hipComplex(deviceGates[i].r11, deviceGates[i].i11)))
                    default: {
                        assert(false);
                    }
                }
            } else {
                assert(deviceGates[i].type == GateType::CZ || deviceGates[i].type == GateType::CU1 || deviceGates[i].type == GateType::CRZ);
                bool isHighBlock = (blockIdx.x >> targetQubit) & 1;
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                int maskControl = (1 << controlQubit) - 1;
                if (!isHighBlock){
                    if (deviceGates[i].type == GateType::CRZ) {
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                            x ^= x >> 3 & 7;
                            RZLo(x, deviceGates[i].r00, -deviceGates[i].i00);
                        }
                    }
                } else {
                    switch (deviceGates[i].type) {
                        case GateType::CZ: {
                            for (int j = threadIdx.x; j < m; j += blockSize) {
                                int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                                x ^= x >> 3 & 7;
                                ZHi(x);
                            }
                            break;    
                        }
                        case GateType::CU1: {
                            for (int j = threadIdx.x; j < m; j += blockSize) {
                                int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                                x ^= x >> 3 & 7;
                                U1Hi(x, make_hipComplex(deviceGates[i].r11, deviceGates[i].i11));
                            }
                            break;
                        }
                        case GateType::CRZ: {
                            for (int j = threadIdx.x; j < m; j += blockSize) {
                                int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                                x ^= x >> 3 & 7;
                                RZHi(x, deviceGates[i].r00, -deviceGates[i].i00);
                            }
                            break;
                        }
                        default: {
                            assert(false);
                        }
                    }
                }
            }
        } else {
            if (controlIsGlobal == 1 && !((blockIdx.x >> controlQubit) & 1)) {
                continue;
            }
            if (!targetIsGlobal) {
                int lo = loArr[(targetQubit * 11) << THREAD_DEP | threadIdx.x];
                int hi = lo ^ (1 << targetQubit) ^ (((1 << targetQubit) >> 3) & 7);
                int add[4];
                if (targetQubit < 8) {
                    add[0] = add[1] = add[2] = 256;
                } else if (targetQubit == 8) {
                    add[0] = 128; add[1] = 384; add[2] = 128;
                } else { // targetQubit == 9
                    add[0] = add[1] = add[2] = 128;
                }
                switch (deviceGates[i].type) {
                    FOLLOW_NEXT(GOC)
                    FOLLOW_NEXT(CU1)
                    CASE_SINGLE(U1, U1Hi(hi, make_hipComplex(deviceGates[i].r11, deviceGates[i].i11)))
                    FOLLOW_NEXT(U2)
                    FOLLOW_NEXT(U)
                    FOLLOW_NEXT(CU)
                    CASE_SINGLE(U3, USingle(lo, hi, make_hipComplex(deviceGates[i].r00, deviceGates[i].i00), make_hipComplex(deviceGates[i].r01, deviceGates[i].i01), make_hipComplex(deviceGates[i].r10, deviceGates[i].i10), make_hipComplex(deviceGates[i].r11, deviceGates[i].i11)));
                    CASE_SINGLE(H, HSingle(lo, hi))
                    FOLLOW_NEXT(X)
                    FOLLOW_NEXT(CNOT)
                    CASE_SINGLE(CCX, XSingle(lo, hi))
                    FOLLOW_NEXT(Y)
                    CASE_SINGLE(CY, YSingle(lo, hi))
                    FOLLOW_NEXT(Z)
                    CASE_SINGLE(CZ, ZHi(hi))
                    FOLLOW_NEXT(RX)
                    CASE_SINGLE(CRX, RXSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i01))
                    FOLLOW_NEXT(RY)
                    CASE_SINGLE(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                    FOLLOW_NEXT(RZ)
                    CASE_SINGLE(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                    CASE_SINGLE(S, SHi(hi))
                    CASE_SINGLE(SDG, SDGHi(hi))
                    CASE_SINGLE(T, THi(hi))
                    CASE_SINGLE(TDG, TDGHi(hi))
                    CASE_SINGLE(GII, GIISingle(lo, hi))
                    CASE_SINGLE(GZZ, GZZSingle(lo, hi))
                    CASE_SINGLE(GCC, GCCSingle(lo, hi, make_hipComplex(deviceGates[i].r00, deviceGates[i].i00)))
                    ID_BREAK()
                    default: {
                        assert(false);
                    }
                }
            } else {
                bool isHighBlock = (blockIdx.x >> targetQubit) & 1;
                switch (deviceGates[i].type) {
                    FOLLOW_NEXT(RZ)
                    CASE_LO_HI(CRZ, RZLo(j, deviceGates[i].r00, -deviceGates[i].i00), RZHi(j, deviceGates[i].r00, -deviceGates[i].i00))
                    FOLLOW_NEXT(Z)
                    CASE_SKIPLO_HI(CZ, ZHi(j))
                    CASE_SKIPLO_HI(S, SHi(j))
                    CASE_SKIPLO_HI(SDG, SDGHi(j))
                    CASE_SKIPLO_HI(T, THi(j))
                    CASE_SKIPLO_HI(TDG, TDGHi(j))
                    FOLLOW_NEXT(GOC)
                    FOLLOW_NEXT(CU1)
                    CASE_SKIPLO_HI(U1, U1Hi(j, make_hipComplex(deviceGates[i].r11, deviceGates[i].i11)))
                    LOHI_SAME(GII, GII(j))
                    LOHI_SAME(GZZ, GZZ(j))
                    LOHI_SAME(GCC, GCC(j, make_hipComplex(deviceGates[i].r00, deviceGates[i].i00)))
                    ID_BREAK()
                    default: {
                        assert(false);
                    }
                }
            }
        }
        __syncthreads();
    }
}

__device__ void fetchData(cuCpx* a, unsigned int* threadBias, unsigned int idx, unsigned int blockHot, unsigned int enumerate, int numLocalQubits) {
    if (threadIdx.x == 0) {
        int bid = blockIdx.x;
        unsigned int bias = 0;
        for (unsigned int bit = 1; bit < (1u << numLocalQubits); bit <<= 1) {
            if (blockHot & bit) {
                if (bid & 1)
                    bias |= bit;
                bid >>= 1;
            }
        }
        blockBias = bias;
    }
    __syncthreads();
    unsigned int bias = blockBias | threadBias[threadIdx.x];
    int x;
    unsigned int y;
    for (x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | threadIdx.x, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
        
        shm[x ^ (x >> 3 & 7)] = a[bias | y];
    }
}

__device__ void saveData(cuCpx* a, unsigned int* threadBias, unsigned int enumerate) {
    unsigned int bias = blockBias | threadBias[threadIdx.x];
    int x;
    unsigned y;
    for (x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | threadIdx.x, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
        
        a[bias | y] = shm[x ^ (x >> 3 & 7)];
    }
}

template <unsigned int blockSize>
__global__ void run(cuCpx* a, unsigned int* threadBias, int* loArr, int* shiftAt, int numLocalQubits, int numGates, unsigned int blockHot, unsigned int enumerate) {
    unsigned int idx = (unsigned int) blockIdx.x * blockSize + threadIdx.x;
    fetchData(a, threadBias, idx, blockHot, enumerate, numLocalQubits);
    __syncthreads();
    doCompute<blockSize>(numGates, loArr, shiftAt);
    __syncthreads();
    saveData(a, threadBias, enumerate);
}

#if GPU_BACKEND == 1 || GPU_BACKEND == 3 || GPU_BACKEND == 4 || GPU_BACKEND == 5
void initControlIdx() {
    int loIdx_host[10][10][128];
    int shiftAt_host[10][10];
    loIdx_device.resize(MyGlobalVars::localGPUs);
    shiftAt_device.resize(MyGlobalVars::localGPUs);
    for (int i = 0; i < MyGlobalVars::localGPUs; i++) {
        hipSetDevice(i);
        hipMalloc(&loIdx_device[i], sizeof(loIdx_host));
        hipMalloc(&shiftAt_device[i], sizeof(shiftAt_host));
    }
    for (int i = 0; i < 128; i++)
        loIdx_host[0][0][i] = (i << 1) ^ ((i & 4) >> 2);

    for (int i = 0; i < 128; i++)
        loIdx_host[1][1][i] = (((i >> 4) << 5) | (i & 15)) ^ ((i & 2) << 3);

    for (int i = 0; i < 128; i++)
        loIdx_host[2][2][i] = (((i >> 5) << 6) | (i & 31)) ^ ((i & 4) << 3);
    
    for (int q = 3; q < 10; q++)
        for (int i = 0; i < 128; i++)
            loIdx_host[q][q][i] = ((i >> q) << (q + 1)) | (i & ((1 << q) - 1));

    for (int c = 0; c < 10; c++) {
        for (int t = 0; t < 10; t++) {
            if (c == t) continue;
            std::vector<int> a[8];
            for (int i = 0; i < 1024; i++) {
                int p = i ^ ((i >> 3) & 7);
                if ((p >> c & 1) && !(p >> t & 1)) {
                    a[i & 7].push_back(i);
                }
            }
            for (int i = 0; i < 8; i++) {
                if (a[i].size() == 0) {
                    for (int j = i + 1; j < 8; j++) {
                        if (a[j].size() == 64) {
                            std::vector<int> tmp = a[j];
                            a[j].clear();
                            for (int k = 0; k < 64; k += 2) {
                                a[i].push_back(tmp[k]);
                                a[j].push_back(tmp[k+1]);
                            }
                            break;
                        }
                    }
                }
            }
            for (int i = 0; i < 128; i++)
                loIdx_host[c][t][i] = a[i & 7][i / 8];
        }
    }

    loIdx_device.resize(MyGlobalVars::localGPUs);
    shiftAt_device.resize(MyGlobalVars::localGPUs);
    for (int g = 0; g < MyGlobalVars::localGPUs; g++) {
        checkCudaErrors(hipMemcpyAsync(loIdx_device[g], loIdx_host[0][0], sizeof(loIdx_host), hipMemcpyHostToDevice, MyGlobalVars::streams[g]));
        checkCudaErrors(hipMemcpyAsync(shiftAt_device[g], shiftAt_host[0], sizeof(shiftAt_host), hipMemcpyHostToDevice, MyGlobalVars::streams[g]));
    }
}
#endif

void copyGatesToSymbol(KernelGate* hostGates, int numGates, hipStream_t& stream, int gpuID) {
    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(deviceGates), hostGates + gpuID * numGates, sizeof(KernelGate) * numGates, 0, hipMemcpyDefault, stream));
}

void launchExecutor(int gridDim, cpx* deviceStateVec, unsigned int* threadBias, int numLocalQubits, int numGates, unsigned int blockHot, unsigned int enumerate, hipStream_t& stream, int gpuID) {
    run<1<<THREAD_DEP><<<gridDim, 1<<THREAD_DEP, 0, stream>>>
        (reinterpret_cast<cuCpx*>(deviceStateVec), threadBias, loIdx_device[gpuID], shiftAt_device[gpuID], numLocalQubits, numGates, blockHot, enumerate);
}