#include "hip/hip_runtime.h"
#include "cuda/kernel.h"
#include <cstdio>
#include <assert.h>
using namespace std;

const int SINGLE_SIZE_DEP = 6; // handle 1 << SINGLE_SIZE_DEP items per thread
const int REDUCE_BLOCK_DEP = 6; // 1 << REDUCE_BLOCK_DEP blocks in final reduction

#define SINGLE_GATE_BEGIN \
    idx_t idx = blockIdx.x * blockSize + threadIdx.x; \
    idx_t mask = (idx_t(1) << targetQubit) - 1; \
    for (idx_t i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        idx_t lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        idx_t hi = lo | (idx_t(1) << targetQubit);

#define SINGLE_GATE_END }

#define CONTROL_GATE_BEGIN \
    idx_t idx = blockIdx.x * blockSize + threadIdx.x; \
    idx_t mask = (idx_t(1) << targetQubit) - 1; \
    for (idx_t i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        idx_t lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        if (!((lo >> controlQubit) & 1)) \
            continue; \
        idx_t hi = lo | (idx_t(1) << targetQubit);

#define CONTROL_GATE_END }

#define CC_GATE_BEGIN \
    idx_t idx = blockIdx.x * blockSize + threadIdx.x; \
    idx_t mask = (idx_t(1) << targetQubit) - 1; \
    for (idx_t i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        idx_t lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        if (!((lo >> c1) & 1)) \
            continue; \
        if (!((lo >> c2) & 1)) \
            continue; \
        idx_t hi = lo | (idx_t(1) << targetQubit);
#define CC_GATE_END }



template <unsigned int blockSize>
__global__ void CCXKernel(cuCpx* a, int numQubit_, int c1, int c2, int targetQubit) {
    CC_GATE_BEGIN {
        value_t real = a[lo].x;
        value_t imag = a[lo].y;
        a[lo].x = a[hi].x;
        a[lo].y = a[hi].y;
        a[hi].x = real;
        a[hi].y = imag;
    } CC_GATE_END
}


template <unsigned int blockSize>
__global__ void CNOTKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        value_t real = a[lo].x;
        value_t imag = a[lo].y;
        a[lo].x = a[hi].x;
        a[lo].y = a[hi].y;
        a[hi].x = real;
        a[hi].y = imag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CYKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = hiImag;
        a[lo].y = -hiReal;
        a[hi].x = -loImag;
        a[hi].y = loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CZKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        a[hi].x = -a[hi].x;
        a[hi].y = -a[hi].y;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CRXKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit, value_t alpha, value_t beta) {
    CONTROL_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = alpha * loReal + beta * hiImag;
        a[lo].y = alpha * loImag - beta * hiReal;
        a[hi].x = alpha * hiReal + beta * loImag;
        a[hi].y = alpha * hiImag - beta * loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CRYKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit, value_t alpha, value_t beta) {
    CONTROL_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = alpha * loReal - beta * hiReal;
        a[lo].y = alpha * loImag - beta * hiImag;
        a[hi].x = beta * loReal + alpha * hiReal;
        a[hi].y = beta * loImag + alpha * hiImag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CRZKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit, value_t alpha, value_t beta) {
    CONTROL_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = alpha * loReal + beta * loImag;
        a[lo].y = alpha * loImag - beta * loReal;
        a[hi].x = alpha * hiReal - beta * hiImag;
        a[hi].y = alpha * hiImag + beta * hiReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CU1Kernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit, value_t alpha, value_t beta) {
    CONTROL_GATE_BEGIN {
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[hi].x = alpha * hiReal - beta * hiImag;
        a[hi].y = alpha * hiImag + beta * hiReal;
    } CONTROL_GATE_END
}

#define COMPLEX_MULTIPLY_REAL(i0, r0, i1, r1) (i0 * i1 - r0 * r1)
#define COMPLEX_MULTIPLY_IMAG(i0, r0, i1, r1) (i0 * r1 + i1 * r0)

template <unsigned int blockSize>
__global__ void CUKernel(cuCpx* a, int numQubit_, int controlQubit, int targetQubit, value_t r00, value_t i00, value_t r01, value_t i01, value_t r10, value_t i10, value_t r11, value_t i11) {
    CONTROL_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = COMPLEX_MULTIPLY_REAL(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r01, i01);
        a[lo].y = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r01, i01);
        a[hi].x = COMPLEX_MULTIPLY_REAL(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r11, i11);
        a[hi].y = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r11, i11);
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void U1Kernel(cuCpx* a, int numQubit_, int targetQubit, value_t alpha, value_t beta) {
    SINGLE_GATE_BEGIN {
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[hi].x = alpha * hiReal - beta * hiImag;
        a[hi].y = alpha * hiImag + beta * hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void UKernel(cuCpx* a, int numQubit_, int targetQubit, value_t r00, value_t i00, value_t r01, value_t i01, value_t r10, value_t i10, value_t r11, value_t i11) {
    SINGLE_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = COMPLEX_MULTIPLY_REAL(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r01, i01);
        a[lo].y = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r01, i01);
        a[hi].x = COMPLEX_MULTIPLY_REAL(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r11, i11);
        a[hi].y = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r11, i11);
    } SINGLE_GATE_END
}

#undef COMPLEX_MULTIPLY_REAL
#undef COMPLEX_MULTIPLY_IMAG

template <unsigned int blockSize>
__global__ void HKernel(cuCpx* a, int numQubit_, int targetQubit, value_t recRoot2) {
    SINGLE_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = recRoot2 * (loReal + hiReal);
        a[lo].y = recRoot2 * (loImag + hiImag);
        a[hi].x = recRoot2 * (loReal - hiReal);
        a[hi].y = recRoot2 * (loImag - hiImag);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void XKernel(cuCpx* a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        value_t real = a[lo].x;
        value_t imag = a[lo].y;
        a[lo].x = a[hi].x;
        a[lo].y = a[hi].y;
        a[hi].x = real;
        a[hi].y = imag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void YKernel(cuCpx* a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = hiImag;
        a[lo].y = -hiReal;
        a[hi].x = -loImag;
        a[hi].y = loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void ZKernel(cuCpx* a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        a[hi].x = -a[hi].x;
        a[hi].y = -a[hi].y;
    } SINGLE_GATE_END
}


template <unsigned int blockSize>
__global__ void SKernel(cuCpx* a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[hi].x = -hiImag;
        a[hi].y = hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void SDGKernel(cuCpx* a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[hi].x = hiImag;
        a[hi].y = -hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void TKernel(cuCpx* a, int numQubit_, int targetQubit, value_t recRoot2) {
    SINGLE_GATE_BEGIN {
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[hi].x = recRoot2 * (hiReal - hiImag);
        a[hi].y = recRoot2 * (hiReal + hiImag);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void TDGKernel(cuCpx* a, int numQubit_, int targetQubit, value_t recRoot2) {
    SINGLE_GATE_BEGIN {
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[hi].x = recRoot2 * (hiReal + hiImag);
        a[hi].y = recRoot2 * (hiImag - hiReal);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void RXKernel(cuCpx* a, int numQubit_, int targetQubit, value_t alpha, value_t beta) {
    SINGLE_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = alpha * loReal + beta * hiImag;
        a[lo].y = alpha * loImag - beta * hiReal;
        a[hi].x = alpha * hiReal + beta * loImag;
        a[hi].y = alpha * hiImag - beta * loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void RYKernel(cuCpx* a, int numQubit_, int targetQubit, value_t alpha, value_t beta) {
    SINGLE_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = alpha * loReal - beta * hiReal;
        a[lo].y = alpha * loImag - beta * hiImag;
        a[hi].x = beta * loReal + alpha * hiReal;
        a[hi].y = beta * loImag + alpha * hiImag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void RZKernel(cuCpx* a, int numQubit_, int targetQubit, value_t alpha, value_t beta) {
    SINGLE_GATE_BEGIN {
        value_t loReal = a[lo].x;
        value_t loImag = a[lo].y;
        value_t hiReal = a[hi].x;
        value_t hiImag = a[hi].y;
        a[lo].x = alpha * loReal + beta * loImag;
        a[lo].y = alpha * loImag - beta * loReal;
        a[hi].x = alpha * hiReal - beta * hiImag;
        a[hi].y = alpha * hiImag + beta * hiReal;
    } SINGLE_GATE_END
}


void kernelExecSimple(cpx* deviceStateVec_, int numQubits, const std::vector<Gate> & gates) {
    cuCpx* deviceStateVec = reinterpret_cast<cuCpx*>(deviceStateVec_);
    checkCudaErrors(hipSetDevice(0));
    int numQubit_ = numQubits - 1;
    int nVec = 1 << numQubit_;
    for (auto& gate: gates) {
        switch (gate.type) {
            case GateType::CCX: {
                CCXKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.controlQubit2, gate.targetQubit);
                break;
            }
            case GateType::CNOT: {
                CNOTKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                break;
            }
            case GateType::CY: {
                CYKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                break;
            }
            case GateType::CZ: {
                CZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                break;
            }
            case GateType::CRX: {
                CRXKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real(), -gate.mat[0][1].imag());
                break;
            }
            case GateType::CRY: {
                CRYKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real(), gate.mat[1][0].imag());
                break;
            }
            case GateType::CU1: {
                CU1Kernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[1][1].real(), gate.mat[1][1].imag());
                break;
            }
            case GateType::CRZ: {
                CRZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real(), - gate.mat[0][0].imag());
                break;
            }
            case GateType::CU: {
                CUKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit,
                    gate.mat[0][0].real(), gate.mat[0][0].imag(),
                    gate.mat[0][1].real(), gate.mat[0][1].imag(),
                    gate.mat[1][0].real(), gate.mat[1][0].imag(),
                    gate.mat[1][1].real(), gate.mat[1][1].imag()
                );
                break;
            }
            case GateType::U1: {
                U1Kernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.targetQubit, gate.mat[1][1].real(), gate.mat[1][1].imag());
                break;
            }
            case GateType::U2: // no break
            case GateType::U:  // no break
            case GateType::U3: {
                UKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.targetQubit,
                    gate.mat[0][0].real(), gate.mat[0][0].imag(),
                    gate.mat[0][1].real(), gate.mat[0][1].imag(),
                    gate.mat[1][0].real(), gate.mat[1][0].imag(),
                    gate.mat[1][1].real(), gate.mat[1][1].imag()
                );
                break;
            }
            case GateType::H: {
                HKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                break;
            }
            case GateType::X: {
                XKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateType::Y: {
                YKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateType::Z: {
                ZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateType::S: {
                SKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateType::SDG: {
                SDGKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateType::T: {
                TKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                break;
            }
            case GateType::TDG: {
                TDGKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                break;
            }
            case GateType::RX: {
                RXKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real(), -gate.mat[0][1].imag());
                break;
            }
            case GateType::RY: {
                RYKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real(), gate.mat[1][0].real());
                break;
            }
            case GateType::RZ: {
                RZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real(), - gate.mat[0][0].imag());
                break;
            }
            default: {
                assert(false);
            }
        }
        checkCudaErrors(hipDeviceSynchronize());
    }
    checkCudaErrors(hipDeviceSynchronize());
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile value_t *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__device__ void blockReduce(volatile value_t *sdata, unsigned int tid) {
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
}

template <unsigned int blockSize>
__global__ void reduce(value_t* g_idata, value_t *g_odata, unsigned int n, unsigned int gridSize) {
    __shared__ value_t sdata[blockSize];
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockSize + threadIdx.x;
    unsigned twoGrid = gridSize << 1;
    sdata[tid] = 0;
    for (int i = idx; i < n; i += twoGrid) {
        sdata[tid] += g_idata[i] + g_idata[i + gridSize];
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void measure(cuCpx* a, value_t* ans, int numQubit_, int targetQubit) {
    __shared__ value_t sdata[blockSize];
    idx_t idx = blockIdx.x * blockSize + threadIdx.x;
    int tid = threadIdx.x;
    idx_t mask = (idx_t(1) << targetQubit) - 1;
    sdata[tid] = 0;
    for (idx_t i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        idx_t lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        sdata[tid] += a[lo].x * a[lo].x + a[lo].y * a[lo].y;
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) ans[blockIdx.x] = sdata[0];
}

value_t kernelMeasure(cpx* deviceStateVec_, int numQubits, int targetQubit) {
    cuCpx* deviceStateVec = reinterpret_cast<cuCpx*>(deviceStateVec_);
    int numQubit_ = numQubits - 1;
    idx_t nVec = 1 << numQubit_;
    idx_t totalBlocks = nVec >> THREAD_DEP >> SINGLE_SIZE_DEP;
    value_t *ans1, *ans2, *ans3;
    checkCudaErrors(hipMalloc(&ans1, sizeof(value_t) * totalBlocks));
    measure<1<<THREAD_DEP><<<totalBlocks, 1<<THREAD_DEP>>>(deviceStateVec, ans1, numQubit_, targetQubit);
    checkCudaErrors(hipMalloc(&ans2, sizeof(value_t) * (1<<REDUCE_BLOCK_DEP)));
    reduce<1<<THREAD_DEP><<<1<<REDUCE_BLOCK_DEP, 1<<THREAD_DEP>>>
        (ans1, ans2, totalBlocks, 1 << (THREAD_DEP + REDUCE_BLOCK_DEP));
    checkCudaErrors(hipHostMalloc(&ans3, sizeof(value_t) * (1<<REDUCE_BLOCK_DEP)));
    checkCudaErrors(hipMemcpy(ans3, ans2, sizeof(value_t) * (1<<REDUCE_BLOCK_DEP), hipMemcpyDeviceToHost));
    value_t ret = 0;
    for (int i = 0; i < (1<<REDUCE_BLOCK_DEP); i++)
        ret += ans3[i];
    checkCudaErrors(hipFree(ans1));
    checkCudaErrors(hipFree(ans2));
    checkCudaErrors(hipHostFree(ans3));
    return ret;
}

// copied and modified from DMSim project
__global__ void packing_kernel(int dim, int m_gpu, int n_qubits, int lg2_m_gpu, const cuCpx* __restrict__ src, cuCpx* __restrict__ dest) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x; 
    for (idx_t i = tid; i < dim * m_gpu; i += blockDim.x*gridDim.x) {
        ////Original version with sementics
        //idx_t w_in_block = i / dim;
        //idx_t block_id = (i % dim) / m_gpu;
        //idx_t h_in_block = (i % dim) % m_gpu;
        //idx_t id_in_dm = w_in_block*dim+(i%dim);
        //idx_t id_in_buf = block_id * m_gpu * m_gpu + w_in_block * m_gpu + h_in_block;

        //Optimized version
        idx_t w_in_block = (i >> n_qubits);
        idx_t block_id = (i & (dim-1)) >> (lg2_m_gpu);
        idx_t h_in_block = (i & (dim-1)) & (m_gpu-1);
        idx_t id_in_dm = (w_in_block << (n_qubits))+(i & (dim-1));
        idx_t id_in_buf = (block_id << (lg2_m_gpu+lg2_m_gpu)) 
            + (w_in_block << (lg2_m_gpu)) + h_in_block;

        dest[id_in_buf] = src[id_in_dm];
    }
}

// copied and modified from DMSim project
__global__ void unpacking_kernel(idx_t dim, idx_t m_gpu, int n_qubits, int lg2_m_gpu, const cuCpx* __restrict__ src, cuCpx* __restrict__ dest) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x; 
    for (idx_t i = tid; i < (dim) * (m_gpu); i+=blockDim.x*gridDim.x) {
        ////Original version with sementics
        //idx_t j = i / dim; 
        //idx_t id_in_buf = j * dim + (i % dim);
        //idx_t block_id = id_in_buf / (m_gpu*m_gpu);
        //idx_t in_block_id = id_in_buf % (m_gpu*m_gpu);
        //idx_t w_in_block = in_block_id / m_gpu;
        //idx_t h_in_block = in_block_id % m_gpu;
        //idx_t dm_w = w_in_block;
        //idx_t dm_h = h_in_block + m_gpu*block_id;
        //idx_t id_in_dim = dm_w * dim + dm_h;

        //Optimized version
        idx_t j = (i >> (n_qubits));
        idx_t id_in_buf = (j << (n_qubits)) + (i & (dim-0x1));
        idx_t block_id = (id_in_buf >> (lg2_m_gpu+lg2_m_gpu));
        idx_t in_block_id = (id_in_buf & ((m_gpu)*(m_gpu)-0x1));
        idx_t w_in_block = (in_block_id >> (lg2_m_gpu));
        idx_t h_in_block = (in_block_id & (m_gpu-1));
        idx_t dm_w = w_in_block;
        idx_t dm_h = h_in_block + (block_id<<(lg2_m_gpu));
        idx_t id_in_dim = (dm_w << (n_qubits)) + dm_h;

        dest[id_in_dim] = src[id_in_buf];
    }
}

#define TRANSPOSE_TILE 16

// copied and modified from DMSim project
__global__ void block_transpose_kernel(idx_t dim, idx_t m_gpu, int n_qubits, int lg2_m_gpu, int n_gpus, const cuCpx* __restrict__ src, cuCpx* __restrict__ dest) {
    __shared__ cuCpx smem[TRANSPOSE_TILE][TRANSPOSE_TILE+1];
    idx_t tlx = threadIdx.x % TRANSPOSE_TILE;
    idx_t tly = threadIdx.x / TRANSPOSE_TILE;
    idx_t n_tile = (m_gpu + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE;
    for (idx_t bid = blockIdx.x; bid < n_tile * n_tile * n_gpus; bid += gridDim.x) {
        idx_t bz = bid / (n_tile * n_tile); 
        idx_t by = bid % (n_tile * n_tile) / n_tile;
        idx_t bx = bid % n_tile;
        idx_t tx = bx * TRANSPOSE_TILE + tlx;
        idx_t ty = by * TRANSPOSE_TILE + tly;

        if (tlx < m_gpu && tly < m_gpu)
        {
            idx_t in_idx = ty * dim + bz * m_gpu + tx;
            cuCpx val = src[in_idx];
            val.y = -val.y;
            smem[tly][tlx] = val;
        }
        __syncthreads();
        if (tlx < m_gpu && tly < m_gpu)
        {
            idx_t out_idx = (bx * TRANSPOSE_TILE + tly) * dim + bz * m_gpu + by * TRANSPOSE_TILE + tlx;
            dest[out_idx] = smem[tlx][tly];
        }
    } 
}

void packing(int numQubits, const cpx* src_, cpx* dest_) {
    const cuCpx* src = reinterpret_cast<const cuCpx*>(src_);
    cuCpx* dest = reinterpret_cast<cuCpx*>(dest_);
    int n2 = numQubits / 2;
    int n_thread = 1 << numQubits >> MyGlobalVars::bit;
    packing_kernel<<<n_thread / 256, 256>>>(1ll << n2, 1ll << (n2 - MyGlobalVars::bit), n2, n2 - MyGlobalVars::bit, src, dest);
}

void unpacking(int numQubits, cpx* src_, cpx* buffer_) {
    cuCpx* src = reinterpret_cast<cuCpx*>(src_);
    cuCpx* buffer = reinterpret_cast<cuCpx*>(buffer_);
    int n2 = numQubits / 2;
    int n_thread = 1 << numQubits >> MyGlobalVars::bit;
    unpacking_kernel<<<n_thread / 256, 256>>>(1ll << n2, 1ll << (n2 - MyGlobalVars::bit), n2, n2 - MyGlobalVars::bit, src, buffer);
    block_transpose_kernel<<<n_thread / TRANSPOSE_TILE / TRANSPOSE_TILE, TRANSPOSE_TILE * TRANSPOSE_TILE>>>(1ll << n2, 1ll << (n2 - MyGlobalVars::bit), n2, n2 - MyGlobalVars::bit, MyGlobalVars::numGPUs, buffer, src);
}
