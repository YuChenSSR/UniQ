#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <assert.h>
using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

const int SINGLE_SIZE_DEP = 0; // handle 1 << SINGLE_SIZE_DEP items per thread
const int THREAD_DEP = 7; // 1 << THREAD_DEP threads per block
const int REDUCE_BLOCK_DEP = 6; // 1 << REDUCE_BLOCK_DEP blocks in final reduction

void kernelInit(ComplexArray& deviceStateVec, int numQubits) {
    assert(numQubits > (SINGLE_SIZE_DEP +THREAD_DEP + 1 + REDUCE_BLOCK_DEP + THREAD_DEP + 1));
    assert(numQubits < 31);
    size_t size = sizeof(qreal) << numQubits;
    checkCudaErrors(hipMalloc(&deviceStateVec.real, size));
    checkCudaErrors(hipMalloc(&deviceStateVec.imag, size));
    checkCudaErrors(hipMemset(deviceStateVec.real, 0, size));
    checkCudaErrors(hipMemset(deviceStateVec.imag, 0, size));
    qreal one = 1;
    checkCudaErrors(hipMemcpy(deviceStateVec.real, &one, sizeof(qreal), hipMemcpyHostToDevice)); // state[0] = 1
}


#define SINGLE_GATE_BEGIN \
    qindex idx = blockIdx.x * blockSize + threadIdx.x; \
    qindex mask = (qindex(1) << targetQubit) - 1; \
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        qindex hi = lo | (qindex(1) << targetQubit);

#define SINGLE_GATE_END }

#define CONTROL_GATE_BEGIN \
    qindex idx = blockIdx.x * blockSize + threadIdx.x; \
    qindex mask = (qindex(1) << targetQubit) - 1; \
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        if (!((lo >> controlQubit) & 1)) \
            continue; \
        qindex hi = lo | (qindex(1) << targetQubit);

#define CONTROL_GATE_END }

template <unsigned int blockSize>
__global__ void controlledNotGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        qreal real = a.real[lo];
        qreal imag = a.imag[lo];
        a.real[lo] = a.real[hi];
        a.imag[lo] = a.imag[hi];
        a.real[hi] = real;
        a.imag[hi] = imag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void controlledPauliYGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = hiImag;
        a.imag[lo] = -hiReal;
        a.real[hi] = -loImag;
        a.imag[hi] = loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void controlAlphaBetaGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, Complex alpha, Complex beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha.real * loReal - alpha.imag * loImag - beta.real * hiReal - beta.imag * hiImag;
        a.imag[lo] = alpha.real * loImag + alpha.imag * loReal - beta.real * hiImag + beta.imag * hiReal;
        a.real[hi] = beta.real * loReal - beta.imag * loImag + alpha.real * hiReal + alpha.imag * hiImag;
        a.imag[hi] = beta.real * loImag + beta.imag * loReal + alpha.real * hiImag - alpha.imag * hiReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void alphaBetaGate(ComplexArray a, int numQubit_, int targetQubit, Complex alpha, Complex beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha.real * loReal - alpha.imag * loImag - beta.real * hiReal - beta.imag * hiImag;
        a.imag[lo] = alpha.real * loImag + alpha.imag * loReal - beta.real * hiImag + beta.imag * hiReal;
        a.real[hi] = beta.real * loReal - beta.imag * loImag + alpha.real * hiReal + alpha.imag * hiImag;
        a.imag[hi] = beta.real * loImag + beta.imag * loReal + alpha.real * hiImag - alpha.imag * hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void hadamardGate(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = recRoot2 * (loReal + hiReal);
        a.imag[lo] = recRoot2 * (loImag + hiImag);
        a.real[hi] = recRoot2 * (loReal - hiReal);
        a.imag[hi] = recRoot2 * (loImag - hiImag);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void pauliXGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal real = a.real[lo];
        qreal imag = a.imag[lo];
        a.real[lo] = a.real[hi];
        a.imag[lo] = a.imag[hi];
        a.real[hi] = real;
        a.imag[hi] = imag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void pauliYGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = hiImag;
        a.imag[lo] = -hiReal;
        a.real[hi] = -loImag;
        a.imag[hi] = loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void pauliZGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        a.real[hi] = -a.real[hi];
        a.imag[hi] = -a.imag[hi];
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void sGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = -hiImag;
        a.imag[hi] = hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void tGate(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = recRoot2 * (hiReal - hiImag);
        a.imag[hi] = recRoot2 * (hiReal + hiImag);
    } SINGLE_GATE_END
}

enum GateImpl {
    GateImplCNot,
    GateImplCAlphaBeta,
    GateImplCPauliY,
    GateImplAlphaBeta,
    GateImplHadamard,
    GateImplPauliX,
    GateImplPauliY,
    GateImplPauliZ,
    GateImplS,
    GateImplT
};

GateImpl toImpl(GateType type) {
    switch (type) {
        case GateHadamard: return GateImplHadamard;
        case GateCNot: return GateImplCNot;
        case GateCPauliY: return GateImplCPauliY;
        case GateCRotateX: return GateImplCAlphaBeta;
        case GateCRotateY: return GateImplCAlphaBeta;
        case GateCRotateZ: return GateImplCAlphaBeta;
        case GatePauliX: return GateImplPauliX;
        case GatePauliY: return GateImplPauliY;
        case GatePauliZ: return GateImplPauliZ;
        case GateRotateX: return GateImplAlphaBeta;
        case GateRotateY: return GateImplAlphaBeta;
        case GateRotateZ: return GateImplAlphaBeta;
        case GateS: return GateImplS;
        case GateT: return GateImplT;
        default: assert(false);
    }
    // shouldn't reach here, just for compile
    return GateImplCNot;
}

void kernelExec(ComplexArray& deviceStateVec, int numQubits, const vector<Gate>& gates) {
    int numQubit_ = numQubits - 1;
    int nVec = 1 << numQubit_;
    for (auto gate: gates) {
        switch (toImpl(gate.type)) {
            case GateImplCNot: {
                controlledNotGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                break;
            }
            case GateImplCPauliY: {
                controlledPauliYGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                break;
            }
            case GateImplCAlphaBeta: {
                controlAlphaBetaGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0], gate.mat[1][0]);
                break;
            }
            case GateImplAlphaBeta: {
                alphaBetaGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0], gate.mat[1][0]);
                break;
            }
            case GateImplHadamard: {
                hadamardGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                break;
            }
            case GateImplPauliX: {
                pauliXGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateImplPauliY: {
                pauliYGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateImplPauliZ: {
                pauliZGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateImplS: {
                sGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                break;
            }
            case GateImplT: {
                tGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                break;
            }
            default: {
                assert(false);
            }
        }
    }
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__device__ void blockReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
}

template <unsigned int blockSize>
__global__ void reduce(qreal* g_idata, qreal *g_odata, unsigned int n, unsigned int gridSize) {
    __shared__ qreal sdata[blockSize];
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockSize + threadIdx.x;
    unsigned twoGrid = gridSize << 1;
    sdata[tid] = 0;
    for (int i = idx; i < n; i += twoGrid) {
        sdata[tid] += g_idata[i] + g_idata[i + gridSize];
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void measure(ComplexArray a, qreal* ans, int numQubit_, int targetQubit) {
    __shared__ qreal sdata[blockSize];
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    int tid = threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    sdata[tid] = 0;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        sdata[tid] += loReal * loReal + loImag * loImag;
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) ans[blockIdx.x] = sdata[0];
}

qreal kernelMeasure(ComplexArray& deviceStateVec, int numQubits, int targetQubit) {
    int numQubit_ = numQubits - 1;
    qindex nVec = 1 << numQubit_;
    qindex totalBlocks = nVec >> THREAD_DEP >> SINGLE_SIZE_DEP;
    qreal *ans1, *ans2, *ans3;
    checkCudaErrors(hipMalloc(&ans1, sizeof(qreal) * totalBlocks));
    measure<1<<THREAD_DEP><<<totalBlocks, 1<<THREAD_DEP>>>(deviceStateVec, ans1, numQubit_, targetQubit);
    checkCudaErrors(hipMalloc(&ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    reduce<1<<THREAD_DEP><<<1<<REDUCE_BLOCK_DEP, 1<<THREAD_DEP>>>
        (ans1, ans2, totalBlocks, 1 << (THREAD_DEP + REDUCE_BLOCK_DEP));
    checkCudaErrors(hipHostMalloc(&ans3, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    checkCudaErrors(hipMemcpy(ans3, ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP), hipMemcpyDeviceToHost));
    qreal ret = 0;
    for (int i = 0; i < (1<<REDUCE_BLOCK_DEP); i++)
        ret += ans3[i];
    checkCudaErrors(hipFree(ans1));
    checkCudaErrors(hipFree(ans2));
    checkCudaErrors(hipHostFree(ans3));
    return ret;
}

Complex kernelGetAmp(ComplexArray& deviceStateVec, qindex idx) {
    Complex ret;
    hipMemcpy(&ret.real, deviceStateVec.real + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    hipMemcpy(&ret.imag, deviceStateVec.imag + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    return ret;
}