#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <assert.h>
using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

const int SINGLE_SIZE_DEP = 0; // handle 1 << SINGLE_SIZE_DEP items per thread
const int THREAD_DEP = 6; // 1 << THREAD_DEP threads per block
const int REDUCE_BLOCK_DEP = 6; // 1 << REDUCE_BLOCK_DEP blocks in final reduction

void kernelInit(ComplexArray& deviceStateVec, int numQubits) {
    assert(numQubits > (SINGLE_SIZE_DEP +THREAD_DEP + 1 + REDUCE_BLOCK_DEP + THREAD_DEP + 1));
    assert(numQubits < 31);
    size_t size = sizeof(qreal) << numQubits;
    checkCudaErrors(hipMalloc(&deviceStateVec.real, size));
    checkCudaErrors(hipMalloc(&deviceStateVec.imag, size));
    checkCudaErrors(hipMemset(deviceStateVec.real, 0, size));
    checkCudaErrors(hipMemset(deviceStateVec.imag, 0, size));
    qreal one = 1;
    checkCudaErrors(hipMemcpy(deviceStateVec.real, &one, sizeof(qreal), hipMemcpyHostToDevice)); // state[0] = 1
}

template <unsigned int blockSize>
__global__ void controlledNotGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        if (!((i >> controlQubit) & 1))
            continue;
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qindex hi = lo | (1 << targetQubit);
        qreal real = a.real[lo]; a.real[lo] = a.real[hi]; a.real[hi] = real;
        qreal imag = a.imag[lo]; a.imag[lo] = a.imag[hi]; a.imag[hi] = imag;
    }
}


template <unsigned int blockSize>
__global__ void hadamardGate(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qindex hi = lo | (1 << targetQubit);
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = recRoot2 * (loReal + hiReal);
        a.imag[lo] = recRoot2 * (loImag + hiImag);
        a.real[hi] = recRoot2 * (loReal - hiReal);
        a.imag[hi] = recRoot2 * (loImag - hiImag);
    }
}

template <unsigned int blockSize>
__global__ void controlAlphaBetaGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, Complex alpha, Complex beta) {
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        if (!((i >> controlQubit) & 1))
            continue;
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qindex hi = lo | (1 << targetQubit);
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha.real * loReal - alpha.imag * loImag - beta.real * hiReal - beta.imag * hiImag;
        a.imag[lo] = alpha.real * loImag + alpha.imag * loReal - beta.real * hiImag + beta.imag * hiReal;
        a.real[hi] = beta.real * loReal - beta.imag * loImag + alpha.real * hiReal + alpha.imag * hiImag;
        a.imag[hi] = beta.real * loImag + beta.imag * loReal + alpha.real * hiImag - alpha.imag * hiReal;
    }
}

void kernelExec(ComplexArray& deviceStateVec, int numQubits, const vector<Gate>& gates) {
    int numQubit_ = numQubits - 1;
    int nVec = 1 << numQubit_;
    for (auto gate: gates) {
        switch (gate.type) {
            case GateHadamard: {
                hadamardGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real);
                break;
            }
            case GateCAlphaBeta: {
                controlAlphaBetaGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                    deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0], gate.mat[1][0]);
                break;
            }
            case GateCNot: {
                controlledNotGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                break;
            }
            default: {
                assert(false);
            }
        }
    }
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__device__ void blockReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
}

template <unsigned int blockSize>
__global__ void reduce(qreal* g_idata, qreal *g_odata, unsigned int n, unsigned int gridSize) {
    __shared__ qreal sdata[blockSize];
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockSize + threadIdx.x;
    unsigned halfGrid = gridSize >> 1;
    sdata[tid] = 0;
    for (int i = idx; i < n; i += gridSize) {
        sdata[tid] += g_idata[i] + g_idata[i + halfGrid];
        i += gridSize;
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void measure(ComplexArray a, qreal* ans, int numQubit_, int targetQubit) {
    __shared__ qreal sdata[blockSize];
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    int tid = threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    sdata[tid] = 0;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        sdata[tid] += loReal * loReal + loImag * loImag;
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) ans[blockIdx.x] = sdata[0];
}

qreal kernelMeasure(ComplexArray& deviceStateVec, int numQubits, int targetQubit) {
    int numQubit_ = numQubits - 1;
    qindex nVec = 1 << numQubit_;
    qindex totalBlocks = nVec >> THREAD_DEP >> SINGLE_SIZE_DEP;
    qreal *ans1, *ans2, *ans3;
    checkCudaErrors(hipMalloc(&ans1, sizeof(qreal) * totalBlocks));
    measure<1<<THREAD_DEP><<<totalBlocks, 1<<THREAD_DEP>>>(deviceStateVec, ans1, numQubit_, targetQubit);
    checkCudaErrors(hipMalloc(&ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    reduce<1<<THREAD_DEP><<<1<<REDUCE_BLOCK_DEP, 1<<THREAD_DEP>>>
        (ans1, ans2, totalBlocks, totalBlocks >> (THREAD_DEP + REDUCE_BLOCK_DEP - 1));
    checkCudaErrors(hipHostMalloc(&ans3, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    checkCudaErrors(hipMemcpy(ans3, ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP), hipMemcpyDeviceToHost));
    qreal ret = 0;
    for (int i = 0; i < (1<<REDUCE_BLOCK_DEP); i++)
        ret += ans3[i];
    checkCudaErrors(hipFree(ans1));
    checkCudaErrors(hipFree(ans2));
    checkCudaErrors(hipHostFree(ans3));
    return ret;
}

Complex kernelGetAmp(ComplexArray& deviceStateVec, qindex idx) {
    Complex ret;
    hipMemcpy(&ret.real, deviceStateVec.real + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    hipMemcpy(&ret.imag, deviceStateVec.imag + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    return ret;
}