#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <cstdio>

__global__ void isnanTest(qComplex *data, int elePerBlock) {
    int l = elePerBlock * blockIdx.x;
    int r = l + elePerBlock;
    for (int i = l + threadIdx.x; i < r; i += blockDim.x) {
        if (isnan(data[i].x) || isnan(data[i].y)) {
            printf("nan at %d\n", i);
            asm("trap;");
        }
    }
}

__global__ void printVector(qComplex *data, int n) { // with gridDim == 1 && blockDim == 1
    for (int i = 0; i < n; i++)
        printf("(%f, %f)", data[i].x, data[i].y);
    printf("\n");
}

void kernelExecBlas(std::vector<qComplex*> deviceStateVec, int numQubits, const Schedule& schedule) {
    assert(MyGlobalVars::numGPUs == 1);
    assert(schedule.localGroups.size() == 1);
    assert(MyGlobalVars::bit == 0);
    int numLocalQubits = numQubits - MyGlobalVars::bit;
    int numElements = 1 << numLocalQubits;
    std::vector<qComplex*> deviceBuffer;
    deviceBuffer.resize(MyGlobalVars::numGPUs);
    for (int g = 0; g < MyGlobalVars::numGPUs; g++) {
        deviceBuffer[g] = deviceStateVec[g] + numElements;        
    }
    hipblasHandle_t handle;
    checkBlasErrors(hipblasCreate(&handle));
    checkBlasErrors(hipblasSetStream(handle, MyGlobalVars::streams[0]));
    auto& fullGroups = schedule.localGroups[0].fullGroups;
    qreal alpha = 1.0, beta = 0.0;
    for (size_t i = 0; i < fullGroups.size(); i++) {
        if (i > 0) {
            checkCuttErrors(cuttExecute(fullGroups[i].cuttPlans[0], deviceStateVec[0], deviceBuffer[0]));
        } else {
            checkCudaErrors(hipMemcpyAsync(deviceBuffer[0], deviceStateVec[0], numElements * sizeof(qComplex), hipMemcpyDeviceToDevice, MyGlobalVars::streams[0]));
        }
        int K = 1 << bitCount(fullGroups[i].relatedQubits);
        // printVector<<<1, 1, 0, MyGlobalVars::streams[0]>>>(fullGroups[i].d[0], K*K);
        // printVector<<<1, 1, 0, MyGlobalVars::streams[0]>>>(deviceBuffer[0], 32);
#ifdef CHECK_NAN_BEFORE_GEMM
        // isnanTest<<<1, 32, 0, MyGlobalVars::streams[0]>>>(fullGroups[i].deviceMats[0], K * K);
        // checkCudaErrors(hipStreamSynchronize(MyGlobalVars::streams[0]));
        // isnanTest<<<numElements / 1024, 32, 0, MyGlobalVars::streams[0]>>>(deviceBuffer[0], 1024);
        // checkCudaErrors(hipStreamSynchronize(MyGlobalVars::streams[0]));
#endif
        checkBlasErrors(cublasGEMM(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            K * 2, numElements / K, K * 2, // M, N, K
            &alpha, fullGroups[i].deviceMats[0], K * 2, // alpha, a, lda
            reinterpret_cast<qreal*>(deviceBuffer[0]), K * 2, // b, ldb
            &beta, reinterpret_cast<qreal*>(deviceStateVec[0]), K * 2 // beta, c, ldc
        ));
        // printVector<<<1, 1, 0, MyGlobalVars::streams[0]>>>(deviceStateVec[0], 32);
    }
    checkCudaErrors(hipStreamSynchronize(MyGlobalVars::streams[0]));
    checkBlasErrors(hipblasDestroy(handle));
}

void kernelMatInit(Schedule& schedule) {
    for (auto& lg: schedule.localGroups) {
        for (int ggID = 0; ggID < lg.fullGroups.size(); ggID ++) {
            auto& gg = lg.fullGroups[ggID];
            int n = 1 << bitCount(gg.relatedQubits);
            qreal realMat[2 * n][2 * n];
            #pragma omp parallel for
            for (int i = 0; i < n; i++)
                for (int j = 0; j < n; j++) {
                    qComplex val = gg.matrix[i * n + j];
                    realMat[i * 2][j * 2] = val.x;
                    realMat[i * 2][j * 2 + 1] = val.y;
                    realMat[i * 2 + 1][j * 2] = -val.y;
                    realMat[i * 2 + 1][j * 2 + 1] = val.x;
                }
            gg.deviceMats.clear();
            for (int g = 0; g < MyGlobalVars::numGPUs; g++) {
                checkCudaErrors(hipSetDevice(g));
                qreal* mat;
                hipMalloc(&mat, n * n * 4 * sizeof(qreal));
                hipMemcpyAsync(mat, realMat, n * n * 4 * sizeof(qreal), hipMemcpyHostToDevice, MyGlobalVars::streams[g]);
                gg.deviceMats.push_back(mat);
            }
        }
    }
}

void kernelMatDestroy(Schedule& schedule) {
    for (int g = 0; g < MyGlobalVars::numGPUs; g++) {
        checkCudaErrors(hipSetDevice(g))
        for (auto& lg: schedule.localGroups) {
            for (auto& gg: lg.fullGroups) {
                checkCudaErrors(hipFree(gg.deviceMats[g]));
            }
        }
    }
    for (auto& lg: schedule.localGroups) {
        for (auto& gg: lg.fullGroups) {
            gg.deviceMats.clear();
        }
    }
}