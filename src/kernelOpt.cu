#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <assert.h>
#include <map>
using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

struct KernelGate {
    int targetQubit;
    int controlQubit;
    GateType type;
    char targetIsGlobal;  // 0-local 1-global
    char controlIsGlobal; // 0-local 1-global 2-not control 
    qreal r00, i00, r01, i01, r10, i10, r11, i11;
};

const int THREAD_DEP = 7; // 1 << THREAD_DEP threads per block
const int MAX_GATE = 152;
const int MAX_QUBIT = 30;
extern __shared__ qreal real[1<<LOCAL_QUBIT_SIZE];
extern __shared__ qreal imag[1<<LOCAL_QUBIT_SIZE];
extern __shared__ qindex blockBias;

__device__ __constant__ double recRoot2 = 0.70710678118654752440084436210485; // more elegant way?
__constant__ KernelGate deviceGates[MAX_GATE];


__device__ inline void XSingle(int lo, int hi) {
    qreal Real = real[lo];
    qreal Imag = imag[lo];
    real[lo] = real[hi];
    imag[lo] = imag[hi];
    real[hi] = Real;
    imag[hi] = Imag;
}

__device__ inline void YSingle(int lo, int hi) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = hiImag;
    imag[lo] = -hiReal;
    real[hi] = -loImag;
    imag[hi] = loReal;
}

__device__ inline void ZHi(int hi) {
    real[hi] = -real[hi];
    imag[hi] = -imag[hi];
}


__device__ inline void RXSingle(int lo, int hi, qreal alpha, qreal beta) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = alpha * loReal + beta * hiImag;
    imag[lo] = alpha * loImag - beta * hiReal;
    real[hi] = alpha * hiReal + beta * loImag;
    imag[hi] = alpha * hiImag - beta * loReal;
}

__device__ inline void RYSingle(int lo, int hi, qreal alpha, qreal beta) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = alpha * loReal - beta * hiReal;
    imag[lo] = alpha * loImag - beta * hiImag;
    real[hi] = beta * loReal + alpha * hiReal;
    imag[hi] = beta * loImag + alpha * hiImag;
}

__device__ inline void RZSingle(int lo, int hi, qreal alpha, qreal beta){
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = alpha * loReal + beta * loImag;
    imag[lo] = alpha * loImag - beta * loReal;
    real[hi] = alpha * hiReal - beta * hiImag;
    imag[hi] = alpha * hiImag + beta * hiReal;
}

__device__ inline void RZLo(int lo, qreal alpha, qreal beta) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    real[lo] = alpha * loReal + beta * loImag;
    imag[lo] = alpha * loImag - beta * loReal;
}

__device__ inline void RZHi(int hi, qreal alpha, qreal beta){
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = alpha * hiReal - beta * hiImag;
    imag[hi] = alpha * hiImag + beta * hiReal;
}

__device__ inline void U1Hi(int hi, qreal alpha, qreal beta) {
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = alpha * hiReal - beta * hiImag;
    imag[hi] = alpha * hiImag + beta * hiReal;
}

#define COMPLEX_MULTIPLY_REAL(i0, r0, i1, r1) (i0 * i1 - r0 * r1)
#define COMPLEX_MULTIPLY_IMAG(i0, r0, i1, r1) (i0 * r1 + i1 * r0)
__device__ inline void USingle(int lo, int hi, qreal r00, qreal i00, qreal r01, qreal i01, qreal r10, qreal i10, qreal r11, qreal i11) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = COMPLEX_MULTIPLY_REAL(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r01, i01);
    imag[lo] = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r01, i01);
    real[hi] = COMPLEX_MULTIPLY_REAL(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r11, i11);
    real[hi] = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r11, i11);
}

__device__ inline void HSingle(int lo, int hi) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = recRoot2 * (loReal + hiReal);
    imag[lo] = recRoot2 * (loImag + hiImag);
    real[hi] = recRoot2 * (loReal - hiReal);
    imag[hi] = recRoot2 * (loImag - hiImag);
}

__device__ inline void SHi(int hi) {
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = -hiImag;
    imag[hi] = hiReal;
}

__device__ inline void THi(int hi) {
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = recRoot2 * (hiReal - hiImag);
    imag[hi] = recRoot2 * (hiReal + hiImag);
}

template <unsigned int blockSize>
__device__ void doCompute(int numGates) {
    for (int i = 0; i < numGates; i++) {
        int controlQubit = deviceGates[i].controlQubit;
        int targetQubit = deviceGates[i].targetQubit;
        char controlIsGlobal = deviceGates[i].controlIsGlobal;
        char targetIsGlobal = deviceGates[i].targetIsGlobal;
        if (!controlIsGlobal) {
            if (!targetIsGlobal) {
                int m = 1 << (LOCAL_QUBIT_SIZE - 2);
                int maskTarget = (1 << targetQubit) - 1;
                int maskControl = (1 << controlQubit) - 1;
                for (int j = threadIdx.x; j < m; j += blockSize) {
                    int lo;
                    if (controlQubit > targetQubit) {
                        lo = ((j >> targetQubit) << (targetQubit + 1)) | (j & maskTarget);
                        lo = ((lo >> controlQubit) << (controlQubit + 1)) | (lo & maskControl) | (1 << controlQubit);
                    } else {
                        lo = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                        lo = ((lo >> targetQubit) << (targetQubit + 1)) | (lo & maskTarget);
                    }
                    int hi = lo | (1 << targetQubit);
                    switch (deviceGates[i].type) {
                        // controlled gates' base type
                        case GateType::CNOT: {
                            XSingle(lo, hi);
                            break;
                        }
                        case GateType::CY: {
                            YSingle(lo, hi);
                            break;
                        }
                        case GateType::CZ: {
                            ZHi(hi);
                            break;
                        }
                        case GateType::CRX: {
                            RXSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00);
                            break;
                        }
                        case GateType::CRY: {
                            RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10);
                            break;
                        }
                        case GateType::CRZ: {
                            RZSingle(lo, hi, deviceGates[i].r00, deviceGates[i].i00);
                            break;
                        }
                        default: {
                            assert(false);
                        }
                    }
                }
            } else {
                assert(deviceGates[i].type == GateType::CZ || deviceGates[i].type == GateType::CRZ);
                bool isHighBlock = (blockIdx.x >> targetQubit) & 1;
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                int maskControl = (1 << controlQubit) - 1;
                if (!isHighBlock){
                    for (int j = threadIdx.x; j < m; j += blockSize) {
                        int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                        if (deviceGates[i].type == GateType::CRZ) {
                            RZLo(x, deviceGates[i].r00, - deviceGates[i].i00);
                        }
                    }
                } else {
                    for (int j = threadIdx.x; j < m; j += blockSize) {
                        int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                        if (deviceGates[i].type == GateType::CRZ) {
                            RZHi(x, deviceGates[i].r00, - deviceGates[i].i00);
                        } else {
                            ZHi(x);
                        }
                    }
                }
            }
        } else {
            if (controlIsGlobal == 1 && !((blockIdx.x >> controlQubit) & 1)) {
                continue;
            }
            if (!targetIsGlobal) {
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                int maskTarget = (1 << targetQubit) - 1;
                for (int j = threadIdx.x; j < m; j += blockSize) {
                    int lo = ((j >> targetQubit) << (targetQubit + 1)) | (j & maskTarget);
                    int hi = lo | (1 << targetQubit);
                    switch (deviceGates[i].type) {
                        case GateType::U1: {
                            U1Hi(hi, deviceGates[i].r11, deviceGates[i].i11);
                            break;
                        }
                        case GateType::U2:
                        case GateType::U3: {
                            USingle(lo, hi, deviceGates[i].r00, deviceGates[i].i00, deviceGates[i].r01, deviceGates[i].i01, deviceGates[i].r10, deviceGates[i].i10, deviceGates[i].r11, deviceGates[i].i11);
                            break;
                        }
                        case GateType::H: {
                            HSingle(lo, hi);
                            break;
                        }
                        case GateType::X: {
                            XSingle(lo, hi);
                            break;
                        }
                        case GateType::Y: {
                            YSingle(lo, hi);
                            break;
                        }
                        case GateType::Z: // no break
                        case GateType::CZ: {
                            ZHi(hi);
                            break;
                        }
                        case GateType::RX: {
                            RXSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i01);
                            break;
                        }
                        case GateType::RY: {
                            RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10);
                            break;
                        }
                        case GateType::RZ: // no break
                        case GateType::CRZ: {
                            RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00);
                            break;
                        }
                        case GateType::S: {
                            SHi(hi);
                            break;
                        }
                        case GateType::T: {
                            THi(hi);
                            break;
                        }
                        default: {
                            assert(false);
                        }
                    }
                }
            } else {
                bool isHighBlock = (blockIdx.x >> targetQubit) & 1;
                switch (deviceGates[i].type) {
                    case GateType::CZ: // no break
                    case GateType::Z: {
                        int m = 1 << LOCAL_QUBIT_SIZE;
                        if (!isHighBlock){
                            for (int j = threadIdx.x; j < m; j += blockSize) {
                                RZLo(j, deviceGates[i].i00, - deviceGates[i].r00);
                            }
                        } else {
                            for (int j = threadIdx.x; j < m; j += blockSize) {
                                RZHi(j, deviceGates[i].i00, - deviceGates[i].r00);
                            }
                        }
                        break;
                    }
                    case GateType::RZ:
                    case GateType::CRZ: {
                        if (!isHighBlock) continue;
                        int m = 1 << LOCAL_QUBIT_SIZE;
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            ZHi(j);
                        }
                        break;
                    }
                    case GateType::S: {
                        if (!isHighBlock) continue;
                        int m = 1 << LOCAL_QUBIT_SIZE;
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            SHi(j);
                        }
                        break;
                    }
                    case GateType::T: {
                        if (!isHighBlock) continue;
                        int m = 1 << LOCAL_QUBIT_SIZE;
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            THi(j);
                        }
                        break;
                    }
                    case GateType::U1: {
                        if (!isHighBlock) continue;
                        int m = 1 << LOCAL_QUBIT_SIZE;
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            U1Hi(j, deviceGates[i].r11, deviceGates[i].i11);
                        }
                    }
                    default: {
                        assert(false);
                    }
                }
            }
        }
        __syncthreads();
    }
}

__device__ void fetchData(ComplexArray a, qindex* threadBias,  qindex idx, qindex blockHot, qindex enumerate, int numQubits) {
    if (threadIdx.x == 0) {
        int bid = blockIdx.x;
        qindex bias = 0;
        for (qindex bit = 1; bit < (qindex(1) << numQubits); bit <<= 1) {
            if (blockHot & bit) {
                if (bid & 1)
                    bias |= bit;
                bid >>= 1;
            }
        }
        blockBias = bias;
    }
    __syncthreads();
    qindex bias = blockBias | threadBias[threadIdx.x];
    for (int x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | threadIdx.x, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
            
        real[x] = a.real[bias | y];
        imag[x] = a.imag[bias | y];
    }
}

__device__ void saveData(ComplexArray a, qindex* threadBias, qindex enumerate) {
    qindex bias = blockBias | threadBias[threadIdx.x];
    for (int x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | threadIdx.x, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
        
        a.real[bias | y] = real[x];
        a.imag[bias | y] = imag[x];
    }
}

#define REDUCE_QUBIT_STEP(x) {\
if (blockSize >= x * 2) { \
    if (tid < x) { \
            sdata[tid] += sdata[tid + x]; \
    } \
} \
__syncthreads();\
}

#define REDUCE_SINGLE_STEP(x) {\
if (blockSize >= x * 2) { \
    if (tid < x) { \
        sdata[tid] += sdata[tid + x]; \
    } \
} \
__syncthreads(); \
}

template <unsigned int blockSize>
__device__ void measure(qreal* result, qindex* threadBias, int numQubits, qindex enumerate) {
    int tid = threadIdx.x;
    qindex bias = blockBias | threadBias[tid];
    __shared__ qreal sdata[blockSize];
    for (int x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | tid, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
            real[x] = real[x] * real[x] + imag[x] * imag[x];
    }
    for (int j = 0; j < numQubits; j++) {
        sdata[tid] = 0;
        for (int x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | tid, y = enumerate;
            x >= 0;
            x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
            
            qindex target = bias | y;
            if ((target >> j) & 1) {
                sdata[tid] += real[x];
            }
        }
        __syncthreads();
        REDUCE_QUBIT_STEP(512);
        REDUCE_QUBIT_STEP(256);
        REDUCE_QUBIT_STEP(128);
        REDUCE_QUBIT_STEP(64);
        REDUCE_QUBIT_STEP(32);
        REDUCE_QUBIT_STEP(16);
        REDUCE_QUBIT_STEP(8);
        REDUCE_QUBIT_STEP(4);
        REDUCE_QUBIT_STEP(2);
        REDUCE_QUBIT_STEP(1);
        if (tid == 0) {
            result[j * gridDim.x + blockIdx.x] = sdata[0];
        }
    }
}

template <unsigned int blockSize>
__global__ void run(ComplexArray a, qindex* threadBias, int numQubits, int numGates, qindex blockHot, qindex enumerate) {
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    fetchData(a, threadBias, idx, blockHot, enumerate, numQubits);
    __syncthreads();
    doCompute<blockSize>(numGates);
    __syncthreads();
    saveData(a, threadBias, enumerate);
}

template <unsigned int blockSize>
__global__ void runLast(ComplexArray a, qreal* result, qindex* threadBias, int numQubits, int numGates, qindex blockHot, qindex enumerate) {
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    fetchData(a, threadBias, idx, blockHot, enumerate, numQubits);
    __syncthreads();
    doCompute<blockSize>(numGates);
    __syncthreads();
    saveData(a, threadBias, enumerate);
    measure<blockSize>(result, threadBias, numQubits, enumerate); // measure must behind save
}

template <unsigned int blockSize, typename T>
__global__ void reduceSum(T* g_idata, T* g_odata, int n) {
    __shared__ T sdata[blockSize];
    int tid = threadIdx.x;
    qindex i = blockIdx.x * n + tid;
    int gridSize = blockSize * 2;
    sdata[tid] = 0;
    while (i < (blockIdx.x + 1) * n) {
        sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize;
    }
    __syncthreads();
    REDUCE_SINGLE_STEP(512);
    REDUCE_SINGLE_STEP(256);
    REDUCE_SINGLE_STEP(128);
    REDUCE_SINGLE_STEP(64);
    REDUCE_SINGLE_STEP(32);
    REDUCE_SINGLE_STEP(16);
    REDUCE_SINGLE_STEP(8);
    REDUCE_SINGLE_STEP(4);
    REDUCE_SINGLE_STEP(2);
    REDUCE_SINGLE_STEP(1);
    // if (tid == 0) printf("%d: sdata %f\n", blockIdx.x, sdata[0]);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

std::vector<qreal> kernelExecOpt(ComplexArray& deviceStateVec, int numQubits, const Schedule& schedule) {
    assert(numQubits <= MAX_QUBIT);
    qindex hostThreadBias[1 << THREAD_DEP];
    qindex* threadBias;
    checkCudaErrors(hipMalloc(&threadBias, sizeof(hostThreadBias)));
    std::vector<qreal> ret;
    for (size_t g = 0; g < schedule.gateGroups.size(); g++) {
        hipEvent_t start, stop;
        if (MEASURE_STAGE) {
            checkCudaErrors(hipEventCreate(&start));
            checkCudaErrors(hipEventCreate(&stop));
            checkCudaErrors(hipEventRecord(start, 0));
        }
        auto& gates = schedule.gateGroups[g].gates;
        // initialize blockHot, enumerate, threadBias
        qindex relatedQubits = schedule.gateGroups[g].relatedQubits;
        int cnt = bitCount(relatedQubits);
        if (cnt < LOCAL_QUBIT_SIZE) {
            int cnt = bitCount(relatedQubits);
            for (int i = 0; i < LOCAL_QUBIT_SIZE; i++) {
                if (!(relatedQubits & (1 << i))) {
                    cnt++;
                    relatedQubits |= (1 << i);
                    if (cnt == LOCAL_QUBIT_SIZE)
                    break;
                }
            }
        }
        qindex blockHot = (qindex(1) << numQubits) - 1 - relatedQubits;
        qindex enumerate = relatedQubits;
        qindex threadHot = 0;
        for (int i = 0; i < THREAD_DEP; i++) {
            qindex x = enumerate & (-enumerate);
            threadHot += x;
            enumerate -= x;
        }
        assert((threadHot | enumerate) == relatedQubits);
        for (int i = (1 << THREAD_DEP) - 1, j = threadHot; i >= 0; i--, j = threadHot & (j - 1)) {
            hostThreadBias[i] = j;
        }
        checkCudaErrors(hipMemcpy(threadBias, hostThreadBias, sizeof(hostThreadBias), hipMemcpyHostToDevice));
        // printf("related %x blockHot %x enumerate %x hostThreadBias[5] %x\n", relatedQubits, blockHot, enumerate, hostThreadBias[5]);

        // initialize gates
        std::map<int, int> toID;
        int localCnt = 0;
        int globalCnt = 0;
        for (int i = 0; i < numQubits; i++) {
            if (relatedQubits & (qindex(1) << i)) {
                toID[i] = localCnt++;
            } else {
                toID[i] = globalCnt++;
            }
        }
        auto isLocalQubit = [relatedQubits] (int x) {
            return relatedQubits >> x & 1;
        };
        KernelGate hostGates[gates.size()];
        assert(gates.size() < MAX_GATE);
        for (size_t i = 0; i < gates.size(); i++) {
            hostGates[i].r00 = gates[i].mat[0][0].real;
            hostGates[i].i00 = gates[i].mat[0][0].imag;
            hostGates[i].r01 = gates[i].mat[0][1].real;
            hostGates[i].i01 = gates[i].mat[0][1].imag;
            hostGates[i].r10 = gates[i].mat[1][0].real;
            hostGates[i].i10 = gates[i].mat[1][0].imag;
            hostGates[i].r11 = gates[i].mat[1][1].real;
            hostGates[i].i11 = gates[i].mat[1][1].imag;
            if (gates[i].controlQubit == -1) {
                hostGates[i].controlQubit = -1;
                hostGates[i].controlIsGlobal = 2;
            } else {
                hostGates[i].controlQubit = toID[gates[i].controlQubit];
                hostGates[i].controlIsGlobal = 1 - isLocalQubit(gates[i].controlQubit);
            }

            hostGates[i].targetQubit = toID[gates[i].targetQubit];
            hostGates[i].targetIsGlobal = 1 - isLocalQubit(gates[i].targetQubit);
            
            hostGates[i].type = gates[i].type;
        }
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(deviceGates), hostGates, sizeof(hostGates)));

        // execute
        qindex gridDim = (1 << numQubits) >> LOCAL_QUBIT_SIZE;
        if (g == schedule.gateGroups.size() - 1) {
            qreal *deviceResultRaw, *deviceResult;
            // raw format:
            // ---------------------------------------------------
            // | qubit 0 (* gridDim) | qubit 1 (* gridDim) | ...
            // ---------------------------------------------------
            checkCudaErrors(hipMalloc(&deviceResultRaw, gridDim * sizeof(qreal) * numQubits));
            runLast<1<<THREAD_DEP><<<gridDim, 1<<THREAD_DEP>>>
                (deviceStateVec, deviceResultRaw, threadBias, numQubits, gates.size(), blockHot, enumerate);
            checkCudaErrors(hipMalloc(&deviceResult, sizeof(qreal) * numQubits));
            reduceSum<1<<THREAD_DEP><<<numQubits, 1<<THREAD_DEP>>>(deviceResultRaw, deviceResult, gridDim);
            ret.resize(numQubits);
            checkCudaErrors(hipMemcpy(ret.data(), deviceResult, sizeof(qreal) * numQubits, hipMemcpyDeviceToHost));
            checkCudaErrors(hipFree(deviceResult));
            checkCudaErrors(hipFree(deviceResultRaw));
        }else {
            run<1<<THREAD_DEP><<<gridDim, 1<<THREAD_DEP>>>
                (deviceStateVec, threadBias, numQubits, gates.size(), blockHot, enumerate);
        }
        if (MEASURE_STAGE) {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float time;
            hipEventElapsedTime(&time, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            printf("[Group %d] time for %x: %f\n", int(g), relatedQubits, time);
        }
        // printf("Group End\n");
    }
    checkCudaErrors(hipFree(threadBias));
    if (MEASURE_STAGE) {
        checkCudaErrors(hipDeviceSynchronize()); // WARNING: for time measure!
    }
    return ret;
}

