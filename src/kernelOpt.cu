#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <assert.h>
#include <map>
using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file %s, line %i.\n", err, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

struct KernelGate {
    int targetQubit;
    int controlQubit;
    int controlQubit2;
    GateType type;
    char targetIsGlobal;  // 0-local 1-global
    char controlIsGlobal; // 0-local 1-global 2-not control 
    char control2IsGlobal; // 0-local 1-global 2-not control
    qreal r00, i00, r01, i01, r10, i10, r11, i11;
};

const int THREAD_DEP = 7; // 1 << THREAD_DEP threads per block
const int MAX_GATE = 600;
const int MAX_QUBIT = 30;
extern __shared__ qreal real[1<<LOCAL_QUBIT_SIZE];
extern __shared__ qreal imag[1<<LOCAL_QUBIT_SIZE];
extern __shared__ qindex blockBias;

__device__ __constant__ qreal recRoot2 = 0.70710678118654752440084436210485; // more elegant way?
__constant__ KernelGate deviceGates[MAX_GATE];
#ifdef USE_GROUP
int* loIdx;
#endif

__device__ __forceinline__ void XSingle(int lo, int hi) {
    qreal Real = real[lo];
    qreal Imag = imag[lo];
    real[lo] = real[hi];
    imag[lo] = imag[hi];
    real[hi] = Real;
    imag[hi] = Imag;
}

__device__ __forceinline__ void YSingle(int lo, int hi) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = hiImag;
    imag[lo] = -hiReal;
    real[hi] = -loImag;
    imag[hi] = loReal;
}

__device__ __forceinline__ void ZHi(int hi) {
    real[hi] = -real[hi];
    imag[hi] = -imag[hi];
}


__device__ __forceinline__ void RXSingle(int lo, int hi, qreal alpha, qreal beta) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = alpha * loReal + beta * hiImag;
    imag[lo] = alpha * loImag - beta * hiReal;
    real[hi] = alpha * hiReal + beta * loImag;
    imag[hi] = alpha * hiImag - beta * loReal;
}

__device__ __forceinline__ void RYSingle(int lo, int hi, qreal alpha, qreal beta) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = alpha * loReal - beta * hiReal;
    imag[lo] = alpha * loImag - beta * hiImag;
    real[hi] = beta * loReal + alpha * hiReal;
    imag[hi] = beta * loImag + alpha * hiImag;
}

__device__ __forceinline__ void RZSingle(int lo, int hi, qreal alpha, qreal beta){
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = alpha * loReal + beta * loImag;
    imag[lo] = alpha * loImag - beta * loReal;
    real[hi] = alpha * hiReal - beta * hiImag;
    imag[hi] = alpha * hiImag + beta * hiReal;
}

__device__ __forceinline__ void RZLo(int lo, qreal alpha, qreal beta) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    real[lo] = alpha * loReal + beta * loImag;
    imag[lo] = alpha * loImag - beta * loReal;
}

__device__ __forceinline__ void RZHi(int hi, qreal alpha, qreal beta){
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = alpha * hiReal - beta * hiImag;
    imag[hi] = alpha * hiImag + beta * hiReal;
}

__device__ __forceinline__ void U1Hi(int hi, qreal alpha, qreal beta) {
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = alpha * hiReal - beta * hiImag;
    imag[hi] = alpha * hiImag + beta * hiReal;
}

#define COMPLEX_MULTIPLY_REAL(i0, r0, i1, r1) (i0 * i1 - r0 * r1)
#define COMPLEX_MULTIPLY_IMAG(i0, r0, i1, r1) (i0 * r1 + i1 * r0)
__device__ __forceinline__ void USingle(int lo, int hi, qreal r00, qreal i00, qreal r01, qreal i01, qreal r10, qreal i10, qreal r11, qreal i11) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = COMPLEX_MULTIPLY_REAL(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r01, i01);
    imag[lo] = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r01, i01);
    real[hi] = COMPLEX_MULTIPLY_REAL(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r11, i11);
    imag[hi] = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r11, i11);
}

__device__ __forceinline__ void HSingle(int lo, int hi) {
    qreal loReal = real[lo];
    qreal loImag = imag[lo];
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[lo] = recRoot2 * (loReal + hiReal);
    imag[lo] = recRoot2 * (loImag + hiImag);
    real[hi] = recRoot2 * (loReal - hiReal);
    imag[hi] = recRoot2 * (loImag - hiImag);
}

__device__ __forceinline__ void SHi(int hi) {
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = -hiImag;
    imag[hi] = hiReal;
}

__device__ __forceinline__ void THi(int hi) {
    qreal hiReal = real[hi];
    qreal hiImag = imag[hi];
    real[hi] = recRoot2 * (hiReal - hiImag);
    imag[hi] = recRoot2 * (hiReal + hiImag);
}

#define FOLLOW_NEXT(TYPE) \
case GateType::TYPE: // no break

#define CASE_CONTROL(TYPE, OP) \
case GateType::TYPE: { \
    OP; \
    lo += shift; hi += shift; \
    lo ^= shift >> 5; hi ^= shift >> 5; \
    OP; \
    break; \
}

#define CASE_CTR_SMALL_SMALL(TYPE, OP) \
case GateType::TYPE: { \
    OP; \
    lo += shift; hi += shift; \
    OP; \
    break; \
}

#define CASE_SINGLE(TYPE, OP) \
case GateType::TYPE: { \
    for (int task = 0; task < 4; task++) { \
        OP; \
        lo += add[task]; hi += add[task]; \
    } \
    break; \
}


#define CASE_LO_HI(TYPE, OP_LO, OP_HI) \
case GateType::TYPE: { \
    int m = 1 << LOCAL_QUBIT_SIZE; \
    if (!isHighBlock){ \
        for (int k = threadIdx.x; k < m; k += blockSize) { \
            int j = k ^ (k >> 5); \
            OP_LO; \
        } \
    } else { \
        for (int k = threadIdx.x; k < m; k += blockSize) { \
            int j = k ^ (k >> 5); \
            OP_HI; \
        } \
    } \
    break; \
}

#define CASE_SKIPLO_HI(TYPE, OP_HI) \
case GateType::TYPE: { \
    if (!isHighBlock) continue; \
    int m = 1 << LOCAL_QUBIT_SIZE; \
    for (int k = threadIdx.x; k < m; k += blockSize) { \
        int j = k ^ (k >> 5); \
        OP_HI; \
    } \
    break; \
}

template <unsigned int blockSize>
__device__ void doCompute(int numGates, int* loArr) {
    for (int i = 0; i < numGates; i++) {
        int controlQubit = deviceGates[i].controlQubit;
        int targetQubit = deviceGates[i].targetQubit;
        char controlIsGlobal = deviceGates[i].controlIsGlobal;
        char targetIsGlobal = deviceGates[i].targetIsGlobal;
        if (deviceGates[i].type == GateType::CCX) {
            int controlQubit2 = deviceGates[i].controlQubit2;
            int control2IsGlobal = deviceGates[i].control2IsGlobal;
            if (!control2IsGlobal) {
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                assert(!controlIsGlobal && !targetIsGlobal);
                assert(deviceGates[i].type == GateType::CCX);
                int maskTarget = (1 << targetQubit) - 1;
                for (int j = threadIdx.x; j < m; j += blockSize) {
                    int lo = ((j >> targetQubit) << (targetQubit + 1)) | (j & maskTarget);
                    if (!(lo >> controlQubit & 1) || !(lo >> controlQubit2 & 1))
                        continue;
                    int hi = lo | (1 << targetQubit);
                    lo = lo ^ (lo >> 5); hi = hi ^ (hi >> 5);
                    XSingle(lo, hi);
                }
                continue;
            }
            if (control2IsGlobal == 1 && !((blockIdx.x >> controlQubit2) & 1)) {
                continue;
            }
        }
        if (!controlIsGlobal) {
            if (!targetIsGlobal) {
                int smallQubit = controlQubit > targetQubit ? targetQubit : controlQubit;
                int largeQubit = controlQubit > targetQubit ? controlQubit : targetQubit;
                int maskSmall = (1 << smallQubit) - 1;
                int maskLarge = (1 << largeQubit) - 1;
                if (controlQubit < 5 && targetQubit < 5) {
                    int lo = loArr[(controlQubit * 5 + targetQubit) << THREAD_DEP | threadIdx.x];
                    int hi = lo ^ (1 << targetQubit);
                    int shift;
                    if (largeQubit == 4) {
                        if (smallQubit == 3) {
                            shift = 1 << 7;
                        } else {
                            shift = 1 << 8;
                        }
                    } else {
                        shift = 1 << 9;
                    }
                    switch (deviceGates[i].type) {
                        CASE_CTR_SMALL_SMALL(CNOT, XSingle(lo, hi))
                        CASE_CTR_SMALL_SMALL(CY, YSingle(lo, hi))
                        CASE_CTR_SMALL_SMALL(CZ, ZHi(hi))
                        CASE_CTR_SMALL_SMALL(CRX, RXSingle(lo, hi, deviceGates[i].r00, deviceGates[i].i01))
                        CASE_CTR_SMALL_SMALL(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                        CASE_CTR_SMALL_SMALL(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                        default: {
                            assert(false);
                        }
                    }
                } else if (controlQubit >= 5 && targetQubit < 5 && controlQubit - targetQubit != 5) {
                    int tid = threadIdx.x;
                    int smallQubit = controlQubit - 5 > targetQubit ? targetQubit : controlQubit - 5;
                    int largeQubit = controlQubit - 5 > targetQubit ? controlQubit - 5 : targetQubit;
                    int maskSmall = (1 << smallQubit) - 1;
                    int maskLarge = (1 << largeQubit) - 1;
                    int maskTarget = (1 << targetQubit) - 1;
                    int x_id = tid >> 5;
                    x_id = x_id >> smallQubit << (smallQubit + 1) | (x_id & maskSmall);
                    x_id = x_id >> largeQubit << (largeQubit + 1) | (x_id & maskLarge);
                    x_id |= (1 << controlQubit - 5);
                    int y_id = tid & 15;
                    y_id = y_id >> targetQubit << (targetQubit + 1) | (y_id & maskTarget);
                    y_id ^= x_id;
                    int lo = x_id << 5 | y_id;
                    if (tid & (1 << 4)) {
                        lo += 33 << targetQubit;
                    }
                    int hi = lo ^ (1 << targetQubit);
                    int shift;
                    if (largeQubit == 4) {
                        if (smallQubit == 3) {
                            shift = 1 << 7;
                        } else {
                            shift = 1 << 8;
                        }
                    } else {
                        shift = 1 << 9;
                    }
                    switch (deviceGates[i].type) {
                        CASE_CTR_SMALL_SMALL(CNOT, XSingle(lo, hi))
                        CASE_CTR_SMALL_SMALL(CY, YSingle(lo, hi))
                        CASE_CTR_SMALL_SMALL(CZ, ZHi(hi))
                        CASE_CTR_SMALL_SMALL(CRX, RXSingle(lo, hi, deviceGates[i].r00, deviceGates[i].i01))
                        CASE_CTR_SMALL_SMALL(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                        CASE_CTR_SMALL_SMALL(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                        default: {
                            assert(false);
                        }
                    }
                } else if (controlQubit < 5 && targetQubit >= 5 && targetQubit - controlQubit != 5) {
                    int tid = threadIdx.x;
                    int smallQubit = controlQubit > targetQubit - 5 ? targetQubit - 5 : controlQubit;
                    int largeQubit = controlQubit > targetQubit - 5 ? controlQubit : targetQubit - 5;
                    int maskSmall = (1 << smallQubit) - 1;
                    int maskLarge = (1 << largeQubit) - 1;
                    int targetMod = (1 << targetQubit) - 1;
                    int maskTarget = (1 << targetMod) - 1;
                    int maskControl = (1 << controlQubit) - 1;
                    int x_id = tid >> 5;
                    x_id = x_id >> smallQubit << (smallQubit + 1) | (x_id & maskSmall);
                    x_id = x_id >> largeQubit << (largeQubit + 1) | (x_id & maskLarge);
                    int y_id = tid & 15;
                    y_id = y_id >> controlQubit << (controlQubit + 1) | (y_id & maskControl);
                    y_id |= 1 << controlQubit;
                    y_id ^= x_id;
                    int lo = x_id << 5 | y_id;
                    if (tid & (1 << 4)) {
                        lo += 31 << controlQubit;
                    }
                    int hi = lo ^ (1 << targetQubit) ^ (1 << (targetQubit - 5));
                    int shift;
                    if (largeQubit == 4) {
                        if (smallQubit == 3) {
                            shift = 1 << 7;
                        } else {
                            shift = 1 << 8;
                        }
                    } else {
                        shift = 1 << 9;
                    }
                    switch (deviceGates[i].type) {
                        CASE_CTR_SMALL_SMALL(CNOT, XSingle(lo, hi))
                        CASE_CTR_SMALL_SMALL(CY, YSingle(lo, hi))
                        CASE_CTR_SMALL_SMALL(CZ, ZHi(hi))
                        CASE_CTR_SMALL_SMALL(CRX, RXSingle(lo, hi, deviceGates[i].r00, deviceGates[i].i01))
                        CASE_CTR_SMALL_SMALL(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                        CASE_CTR_SMALL_SMALL(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                        default: {
                            assert(false);
                        }
                    }
                } else {
                    int m = 1 << (LOCAL_QUBIT_SIZE - 2);
                    int lo = ((threadIdx.x >> smallQubit) << (smallQubit + 1)) | (threadIdx.x & maskSmall);
                    lo = ((lo >> largeQubit) << (largeQubit + 1)) | (lo & maskLarge);
                    lo |= 1 << controlQubit;
                    int hi = lo | (1 << targetQubit);
                    lo ^= lo >> 5;
                    hi ^= hi >> 5;
                    int shift;
                    if (largeQubit == 9) {
                        if (smallQubit == 8) {
                            shift = 1 << 7;
                        } else {
                            shift = 1 << 8;
                        }
                    } else {
                        shift = 1 << 9;
                    }
                    switch (deviceGates[i].type) {
                        CASE_CONTROL(CNOT, XSingle(lo, hi))
                        CASE_CONTROL(CY, YSingle(lo, hi))
                        CASE_CONTROL(CZ, ZHi(hi))
                        CASE_CONTROL(CRX, RXSingle(lo, hi, deviceGates[i].r00, deviceGates[i].i01))
                        CASE_CONTROL(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                        CASE_CONTROL(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                        default: {
                            assert(false);
                        }
                    }
                }
            } else {
                assert(deviceGates[i].type == GateType::CZ || deviceGates[i].type == GateType::CRZ);
                bool isHighBlock = (blockIdx.x >> targetQubit) & 1;
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                int maskControl = (1 << controlQubit) - 1;
                if (!isHighBlock){
                    if (deviceGates[i].type == GateType::CRZ) {
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                            x ^= x >> 5;
                            RZLo(x, deviceGates[i].r00, - deviceGates[i].i00);
                        }
                    }
                } else {
                    if (deviceGates[i].type == GateType::CRZ) {
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                            x ^= x >> 5;
                            RZHi(x, deviceGates[i].r00, - deviceGates[i].i00);
                        }
                    } else {
                        for (int j = threadIdx.x; j < m; j += blockSize) {
                            int x = ((j >> controlQubit) << (controlQubit + 1)) | (j & maskControl)  | (1 << controlQubit);
                            x ^= x >> 5;
                            ZHi(x);
                        }
                    }
                }
            }
        } else {
            if (controlIsGlobal == 1 && !((blockIdx.x >> controlQubit) & 1)) {
                continue;
            }
            if (!targetIsGlobal) {
                int m = 1 << (LOCAL_QUBIT_SIZE - 1);
                int maskTarget = (1 << targetQubit) - 1;
                int x_id = threadIdx.x >> 5; \
                switch (targetQubit) {
                    case 0: case 5: x_id <<= 1; break;
                    case 1: case 6: x_id = (x_id & 2) << 1 | (x_id & 1); break;
                    default: break;
                }
                int lo, hi;
                if (targetQubit < 5) {
                    int y_id = threadIdx.x & 15;
                    y_id = (y_id >> targetQubit) << (targetQubit + 1) | (y_id & maskTarget);
                    lo = x_id << 5 | y_id;
                    lo += (threadIdx.x & 31) < 16 ? 0 : 33 << targetQubit;
                    hi = lo ^ (1 << targetQubit);
                } else {
                    int y_id = threadIdx.x & 31;
                    lo = x_id << 5 | y_id;
                    hi = lo ^ (33 << (targetQubit - 5));
                }
                int add[4];
                switch (targetQubit) {
                    case 0: case 1: case 2:
                    case 5: case 6: case 7: {
                        add[0] = add[1] = add[2] = 256;
                        break;
                    }
                    case 3: case 8: {
                        add[0] = 128; add[1] = 384; add[2] = 128;
                        break;
                    }
                    case 4: case 9: {
                        add[0] = 128; add[1] = 128; add[2] = 128;
                        break;
                    }
                }
                switch (deviceGates[i].type) {
                    CASE_SINGLE(U1, U1Hi(hi, deviceGates[i].r11, deviceGates[i].i11))
                    FOLLOW_NEXT(U2)
                    CASE_SINGLE(U3, USingle(lo, hi, deviceGates[i].r00, deviceGates[i].i00, deviceGates[i].r01, deviceGates[i].i01, deviceGates[i].r10, deviceGates[i].i10, deviceGates[i].r11, deviceGates[i].i11))
                    CASE_SINGLE(H, HSingle(lo, hi))
                    FOLLOW_NEXT(X)
                    FOLLOW_NEXT(CNOT)
                    CASE_SINGLE(CCX, XSingle(lo, hi))
                    FOLLOW_NEXT(Y)
                    CASE_SINGLE(CY, YSingle(lo, hi))
                    FOLLOW_NEXT(Z)
                    CASE_SINGLE(CZ, ZHi(hi))
                    FOLLOW_NEXT(RX)
                    CASE_SINGLE(CRX, RXSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i01))
                    FOLLOW_NEXT(RY)
                    CASE_SINGLE(CRY, RYSingle(lo, hi, deviceGates[i].r00, deviceGates[i].r10))
                    FOLLOW_NEXT(RZ)
                    CASE_SINGLE(CRZ, RZSingle(lo, hi, deviceGates[i].r00, -deviceGates[i].i00))
                    CASE_SINGLE(S, SHi(hi))
                    CASE_SINGLE(T, THi(hi))
                    default: {
                        assert(false);
                    }
                }
            } else {
                bool isHighBlock = (blockIdx.x >> targetQubit) & 1;
                switch (deviceGates[i].type) {
                    FOLLOW_NEXT(RZ)
                    CASE_LO_HI(CRZ, RZLo(j, deviceGates[i].i00, - deviceGates[i].r00), RZHi(j, deviceGates[i].i00, - deviceGates[i].r00))
                    FOLLOW_NEXT(Z)
                    CASE_SKIPLO_HI(CZ, ZHi(j))
                    CASE_SKIPLO_HI(S, SHi(j))
                    CASE_SKIPLO_HI(T, THi(j))
                    CASE_SKIPLO_HI(U1, U1Hi(j, deviceGates[i].r11, deviceGates[i].i11))
                    default: {
                        assert(false);
                    }
                }
            }
        }
        __syncthreads();
    }
}

__device__ void fetchData(ComplexArray a, qindex* threadBias,  qindex idx, qindex blockHot, qindex enumerate, int numQubits) {
    if (threadIdx.x == 0) {
        int bid = blockIdx.x;
        qindex bias = 0;
        for (qindex bit = 1; bit < (qindex(1) << numQubits); bit <<= 1) {
            if (blockHot & bit) {
                if (bid & 1)
                    bias |= bit;
                bid >>= 1;
            }
        }
        blockBias = bias;
    }
    __syncthreads();
    qindex bias = blockBias | threadBias[threadIdx.x];
    for (int x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | threadIdx.x, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
            
        real[x ^ (x >> 5)] = a.real[bias | y];
        imag[x ^ (x >> 5)] = a.imag[bias | y];
    }
}

__device__ void saveData(ComplexArray a, qindex* threadBias, qindex enumerate) {
    qindex bias = blockBias | threadBias[threadIdx.x];
    for (int x = ((1 << (LOCAL_QUBIT_SIZE - THREAD_DEP)) - 1) << THREAD_DEP | threadIdx.x, y = enumerate;
        x >= 0;
        x -= (1 << THREAD_DEP), y = enumerate & (y - 1)) {
        
        a.real[bias | y] = real[x ^ (x >> 5)];
        a.imag[bias | y] = imag[x ^ (x >> 5)];
    }
}

template <unsigned int blockSize>
__global__ void run(ComplexArray a, qindex* threadBias, int* loArr, int numQubits, int numGates, qindex blockHot, qindex enumerate) {
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    fetchData(a, threadBias, idx, blockHot, enumerate, numQubits);
    __syncthreads();
    doCompute<blockSize>(numGates, loArr);
    __syncthreads();
    saveData(a, threadBias, enumerate);
}

#ifdef USE_GROUP
void initControlIdx() {
    hipMalloc(&loIdx, sizeof(int) * 25 * (1 << THREAD_DEP));
    int loIdx_host[5][5][1 << THREAD_DEP];
    for (int controlQubit = 0; controlQubit < 5; controlQubit ++)
        for (int targetQubit = 0; targetQubit < 5; targetQubit ++)
            for (int tid = 0; tid < (1 << THREAD_DEP); tid++) {
                if (controlQubit == targetQubit) continue;
                int smallQubit = controlQubit > targetQubit ? targetQubit : controlQubit;
                int largeQubit = controlQubit > targetQubit ? controlQubit : targetQubit;
                int maskSmall = (1 << smallQubit) - 1;
                int maskLarge = (1 << largeQubit) - 1;
                int x_id = tid >> 5;
                x_id = x_id >> smallQubit << (smallQubit + 1) | (x_id & maskSmall);
                x_id = x_id >> largeQubit << (largeQubit + 1) | (x_id & maskLarge);
                int y_id = tid & 7;
                y_id = y_id >> smallQubit << (smallQubit + 1) | (y_id & maskSmall);
                y_id = y_id >> largeQubit << (largeQubit + 1) | (y_id & maskLarge);
                y_id |= 1 << controlQubit;
                int lo = x_id << 5 | y_id;
                if (tid & (1 << 3)) {
                    lo += 33 << targetQubit;
                }
                if (tid & (1 << 4)) {
                    lo += 31 << controlQubit;
                }
                loIdx_host[controlQubit][targetQubit][tid] = lo;
            }
    checkCudaErrors(hipMemcpy(loIdx, loIdx_host[0][0], sizeof(int) * 25 * (1 << THREAD_DEP), hipMemcpyHostToDevice));
}
#endif

std::vector<qreal> kernelExecOpt(ComplexArray& deviceStateVec, int numQubits, const Schedule& schedule) {
    assert(numQubits <= MAX_QUBIT);
    qindex hostThreadBias[1 << THREAD_DEP];
    qindex* threadBias;
    checkCudaErrors(hipMalloc(&threadBias, sizeof(hostThreadBias)));
    std::vector<qreal> ret;
    for (size_t g = 0; g < schedule.gateGroups.size(); g++) {
#ifdef MEASURE_STAGE
        hipEvent_t start, stop;
            checkCudaErrors(hipEventCreate(&start));
            checkCudaErrors(hipEventCreate(&stop));
            checkCudaErrors(hipEventRecord(start, 0));
#endif
        auto& gates = schedule.gateGroups[g].gates;
        // initialize blockHot, enumerate, threadBias
        qindex relatedQubits = schedule.gateGroups[g].relatedQubits;
        int cnt = bitCount(relatedQubits);
        if (cnt < LOCAL_QUBIT_SIZE) {
            int cnt = bitCount(relatedQubits);
            for (int i = 0; i < LOCAL_QUBIT_SIZE; i++) {
                if (!(relatedQubits & (1 << i))) {
                    cnt++;
                    relatedQubits |= (1 << i);
                    if (cnt == LOCAL_QUBIT_SIZE)
                        break;
                }
            }
        }
        qindex blockHot = (qindex(1) << numQubits) - 1 - relatedQubits;
        qindex enumerate = relatedQubits;
        qindex threadHot = 0;
        for (int i = 0; i < THREAD_DEP; i++) {
            qindex x = enumerate & (-enumerate);
            threadHot += x;
            enumerate -= x;
        }
        assert((threadHot | enumerate) == relatedQubits);
        for (int i = (1 << THREAD_DEP) - 1, j = threadHot; i >= 0; i--, j = threadHot & (j - 1)) {
            hostThreadBias[i] = j;
        }
        checkCudaErrors(hipMemcpy(threadBias, hostThreadBias, sizeof(hostThreadBias), hipMemcpyHostToDevice));
        // printf("related %x blockHot %x enumerate %x hostThreadBias[5] %x\n", relatedQubits, blockHot, enumerate, hostThreadBias[5]);

        // initialize gates
        std::map<int, int> toID;
        int localCnt = 0;
        int globalCnt = 0;
        for (int i = 0; i < numQubits; i++) {
            if (relatedQubits & (qindex(1) << i)) {
                toID[i] = localCnt++;
            } else {
                toID[i] = globalCnt++;
            }
        }
        auto isLocalQubit = [relatedQubits] (int x) {
            return relatedQubits >> x & 1;
        };
        KernelGate hostGates[gates.size()];
        assert(gates.size() < MAX_GATE);
        for (size_t i = 0; i < gates.size(); i++) {
            hostGates[i].r00 = gates[i].mat[0][0].real;
            hostGates[i].i00 = gates[i].mat[0][0].imag;
            hostGates[i].r01 = gates[i].mat[0][1].real;
            hostGates[i].i01 = gates[i].mat[0][1].imag;
            hostGates[i].r10 = gates[i].mat[1][0].real;
            hostGates[i].i10 = gates[i].mat[1][0].imag;
            hostGates[i].r11 = gates[i].mat[1][1].real;
            hostGates[i].i11 = gates[i].mat[1][1].imag;
            if (gates[i].controlQubit2 != -1) {
                int c1 = gates[i].controlQubit;
                int c2 = gates[i].controlQubit2;
                if (isLocalQubit(c2) && !isLocalQubit(c1)) {
                    int c = c1; c1 = c2; c2 = c;
                }
                hostGates[i].controlQubit2 = toID[c2];
                hostGates[i].control2IsGlobal = 1 - isLocalQubit(c2);
                hostGates[i].controlQubit = toID[c1];
                hostGates[i].controlIsGlobal = 1 - isLocalQubit(c1);
            } else if (gates[i].controlQubit != -1) {
                hostGates[i].controlQubit2 = -1;
                hostGates[i].control2IsGlobal = 2;
                hostGates[i].controlQubit = toID[gates[i].controlQubit];
                hostGates[i].controlIsGlobal = 1 - isLocalQubit(gates[i].controlQubit);
            } else {
                hostGates[i].controlQubit2 = -1;
                hostGates[i].control2IsGlobal = 2;
                hostGates[i].controlQubit = -1;
                hostGates[i].controlIsGlobal = 2;
            }

            hostGates[i].targetQubit = toID[gates[i].targetQubit];
            hostGates[i].targetIsGlobal = 1 - isLocalQubit(gates[i].targetQubit);
            
            hostGates[i].type = gates[i].type;
        }
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(deviceGates), hostGates, sizeof(hostGates)));

        // execute
        qindex gridDim = (1 << numQubits) >> LOCAL_QUBIT_SIZE;
        run<1<<THREAD_DEP><<<gridDim, 1<<THREAD_DEP>>>
            (deviceStateVec, threadBias, loIdx, numQubits, gates.size(), blockHot, enumerate);
#ifdef MEASURE_STAGE
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float time;
            hipEventElapsedTime(&time, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            printf("[Group %d] time for %x: %f\n", int(g), relatedQubits, time);
#endif
        // printf("Group End\n");
    }
    checkCudaErrors(hipFree(threadBias));
    checkCudaErrors(hipDeviceSynchronize()); // WARNING: for time measure!
    return ret;
}

