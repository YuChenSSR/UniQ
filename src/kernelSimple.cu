#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <assert.h>
using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

const int SINGLE_SIZE_DEP = 0; // handle 1 << SINGLE_SIZE_DEP items per thread
const int THREAD_DEP = 7; // 1 << THREAD_DEP threads per block
const int REDUCE_BLOCK_DEP = 6; // 1 << REDUCE_BLOCK_DEP blocks in final reduction

void kernelInit(ComplexArray& deviceStateVec, int numQubits) {
    hipError_t cuda_status;
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
        printf("hipSetDevice failed! ");
        exit(1);
	}
    assert(numQubits < 31);
    size_t size = sizeof(qreal) << numQubits;
    checkCudaErrors(hipMalloc(&deviceStateVec.real, size));
    checkCudaErrors(hipMalloc(&deviceStateVec.imag, size));
    checkCudaErrors(hipMemset(deviceStateVec.real, 0, size));
    checkCudaErrors(hipMemset(deviceStateVec.imag, 0, size));
    qreal one = 1;
    checkCudaErrors(hipMemcpy(deviceStateVec.real, &one, sizeof(qreal), hipMemcpyHostToDevice)); // state[0] = 1
}


#define SINGLE_GATE_BEGIN \
    qindex idx = blockIdx.x * blockSize + threadIdx.x; \
    qindex mask = (qindex(1) << targetQubit) - 1; \
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        qindex hi = lo | (qindex(1) << targetQubit);

#define SINGLE_GATE_END }

#define CONTROL_GATE_BEGIN \
    qindex idx = blockIdx.x * blockSize + threadIdx.x; \
    qindex mask = (qindex(1) << targetQubit) - 1; \
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        if (!((lo >> controlQubit) & 1)) \
            continue; \
        qindex hi = lo | (qindex(1) << targetQubit);

#define CONTROL_GATE_END }

template <unsigned int blockSize>
__global__ void CNOTKernel(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        qreal real = a.real[lo];
        qreal imag = a.imag[lo];
        a.real[lo] = a.real[hi];
        a.imag[lo] = a.imag[hi];
        a.real[hi] = real;
        a.imag[hi] = imag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CYKernel(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = hiImag;
        a.imag[lo] = -hiReal;
        a.real[hi] = -loImag;
        a.imag[hi] = loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CZKernel(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        a.real[hi] = -a.real[hi];
        a.imag[hi] = -a.imag[hi];
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CRXKernel(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, qreal alpha, qreal beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * hiImag;
        a.imag[lo] = alpha * loImag - beta * hiReal;
        a.real[hi] = alpha * hiReal + beta * loImag;
        a.imag[hi] = alpha * hiImag - beta * loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CRYKernel(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, qreal alpha, qreal beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal - beta * hiReal;
        a.imag[lo] = alpha * loImag - beta * hiImag;
        a.real[hi] = beta * loReal + alpha * hiReal;
        a.imag[hi] = beta * loImag + alpha * hiImag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void CRZKernel(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, qreal alpha, qreal beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * loImag;
        a.imag[lo] = alpha * loImag - beta * loReal;
        a.real[hi] = alpha * hiReal - beta * hiImag;
        a.imag[hi] = alpha * hiImag + beta * hiReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void U1Kernel(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = alpha * hiReal - beta * hiImag;
        a.imag[hi] = alpha * hiImag + beta * hiReal;
    } SINGLE_GATE_END
}

#define COMPLEX_MULTIPLY_REAL(i0, r0, i1, r1) (i0 * i1 - r0 * r1)
#define COMPLEX_MULTIPLY_IMAG(i0, r0, i1, r1) (i0 * r1 + i1 * r0)

template <unsigned int blockSize>
__global__ void UKernel(ComplexArray a, int numQubit_, int targetQubit, qreal r00, qreal i00, qreal r01, qreal i01, qreal r10, qreal i10, qreal r11, qreal i11) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = COMPLEX_MULTIPLY_REAL(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r01, i01);
        a.imag[lo] = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r00, i00) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r01, i01);
        a.real[hi] = COMPLEX_MULTIPLY_REAL(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_REAL(hiReal, hiImag, r11, i11);
        a.imag[hi] = COMPLEX_MULTIPLY_IMAG(loReal, loImag, r10, i10) + COMPLEX_MULTIPLY_IMAG(hiReal, hiImag, r11, i11);
    } SINGLE_GATE_END
}

#undef COMPLEX_MULTIPLY_REAL
#undef COMPLEX_MULTIPLY_IMAG

template <unsigned int blockSize>
__global__ void HKernel(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = recRoot2 * (loReal + hiReal);
        a.imag[lo] = recRoot2 * (loImag + hiImag);
        a.real[hi] = recRoot2 * (loReal - hiReal);
        a.imag[hi] = recRoot2 * (loImag - hiImag);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void XKernel(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal real = a.real[lo];
        qreal imag = a.imag[lo];
        a.real[lo] = a.real[hi];
        a.imag[lo] = a.imag[hi];
        a.real[hi] = real;
        a.imag[hi] = imag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void YKernel(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = hiImag;
        a.imag[lo] = -hiReal;
        a.real[hi] = -loImag;
        a.imag[hi] = loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void ZKernel(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        a.real[hi] = -a.real[hi];
        a.imag[hi] = -a.imag[hi];
    } SINGLE_GATE_END
}


template <unsigned int blockSize>
__global__ void SKernel(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = -hiImag;
        a.imag[hi] = hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void TKernel(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = recRoot2 * (hiReal - hiImag);
        a.imag[hi] = recRoot2 * (hiReal + hiImag);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void RXKernel(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * hiImag;
        a.imag[lo] = alpha * loImag - beta * hiReal;
        a.real[hi] = alpha * hiReal + beta * loImag;
        a.imag[hi] = alpha * hiImag - beta * loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void RYKernel(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal - beta * hiReal;
        a.imag[lo] = alpha * loImag - beta * hiImag;
        a.real[hi] = beta * loReal + alpha * hiReal;
        a.imag[hi] = beta * loImag + alpha * hiImag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void RZKernel(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * loImag;
        a.imag[lo] = alpha * loImag - beta * loReal;
        a.real[hi] = alpha * hiReal - beta * hiImag;
        a.imag[hi] = alpha * hiImag + beta * hiReal;
    } SINGLE_GATE_END
}


void kernelExecSimple(ComplexArray& deviceStateVec, int numQubits, const Schedule& schedule) {
    int numQubit_ = numQubits - 1;
    int nVec = 1 << numQubit_;
    for (auto& gg: schedule.gateGroups) {
        for (auto& gate: gg.gates) {
            switch (gate.type) {
                case GateType::CNOT: {
                    CNOTKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                    break;
                }
                case GateType::CY: {
                    CYKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                    break;
                }
                case GateType::CZ: {
                    CZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                    break;
                }
                case GateType::CRX: {
                    CRXKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real, -gate.mat[0][1].imag);
                    break;
                }
                case GateType::CRY: {
                    CRYKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real, gate.mat[1][0].real);
                    break;
                }
                case GateType::CRZ: {
                    CRZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real, - gate.mat[0][0].imag);
                    break;
                }
                case GateType::U1: {
                    U1Kernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[1][1].real, gate.mat[1][1].imag);
                    break;
                }
                case GateType::U2: // no break
                case GateType::U3: {
                    UKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit,
                        gate.mat[0][0].real, gate.mat[0][0].imag,
                        gate.mat[0][1].real, gate.mat[0][1].imag,
                        gate.mat[1][0].real, gate.mat[1][0].imag,
                        gate.mat[1][1].real, gate.mat[1][1].imag
                    );
                    break;
                }
                case GateType::H: {
                    HKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                    break;
                }
                case GateType::X: {
                    XKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GateType::Y: {
                    YKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GateType::Z: {
                    ZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GateType::S: {
                    SKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GateType::T: {
                    TKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                    break;
                }
                case GateType::RX: {
                    RXKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real, -gate.mat[0][1].imag);
                    break;
                }
                case GateType::RY: {
                    RYKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real, gate.mat[1][0].real);
                    break;
                }
                case GateType::RZ: {
                    RZKernel<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real, - gate.mat[0][0].imag);
                    break;
                }
                default: {
                    assert(false);
                }
            }
        }
    }
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__device__ void blockReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
}

template <unsigned int blockSize>
__global__ void reduce(qreal* g_idata, qreal *g_odata, unsigned int n, unsigned int gridSize) {
    __shared__ qreal sdata[blockSize];
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockSize + threadIdx.x;
    unsigned twoGrid = gridSize << 1;
    sdata[tid] = 0;
    for (int i = idx; i < n; i += twoGrid) {
        sdata[tid] += g_idata[i] + g_idata[i + gridSize];
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void measure(ComplexArray a, qreal* ans, int numQubit_, int targetQubit) {
    __shared__ qreal sdata[blockSize];
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    int tid = threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    sdata[tid] = 0;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        sdata[tid] += loReal * loReal + loImag * loImag;
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) ans[blockIdx.x] = sdata[0];
}

qreal kernelMeasure(ComplexArray& deviceStateVec, int numQubits, int targetQubit) {
    int numQubit_ = numQubits - 1;
    qindex nVec = 1 << numQubit_;
    qindex totalBlocks = nVec >> THREAD_DEP >> SINGLE_SIZE_DEP;
    qreal *ans1, *ans2, *ans3;
    checkCudaErrors(hipMalloc(&ans1, sizeof(qreal) * totalBlocks));
    measure<1<<THREAD_DEP><<<totalBlocks, 1<<THREAD_DEP>>>(deviceStateVec, ans1, numQubit_, targetQubit);
    checkCudaErrors(hipMalloc(&ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    reduce<1<<THREAD_DEP><<<1<<REDUCE_BLOCK_DEP, 1<<THREAD_DEP>>>
        (ans1, ans2, totalBlocks, 1 << (THREAD_DEP + REDUCE_BLOCK_DEP));
    checkCudaErrors(hipHostMalloc(&ans3, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    checkCudaErrors(hipMemcpy(ans3, ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP), hipMemcpyDeviceToHost));
    qreal ret = 0;
    for (int i = 0; i < (1<<REDUCE_BLOCK_DEP); i++)
        ret += ans3[i];
    checkCudaErrors(hipFree(ans1));
    checkCudaErrors(hipFree(ans2));
    checkCudaErrors(hipHostFree(ans3));
    return ret;
}

Complex kernelGetAmp(ComplexArray& deviceStateVec, qindex idx) {
    Complex ret;
    hipMemcpy(&ret.real, deviceStateVec.real + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    hipMemcpy(&ret.imag, deviceStateVec.imag + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    return ret;
}