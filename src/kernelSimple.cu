#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <assert.h>
using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
    if (hipSuccess != err)
    {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

const int SINGLE_SIZE_DEP = 0; // handle 1 << SINGLE_SIZE_DEP items per thread
const int THREAD_DEP = 7; // 1 << THREAD_DEP threads per block
const int REDUCE_BLOCK_DEP = 6; // 1 << REDUCE_BLOCK_DEP blocks in final reduction

void kernelInit(ComplexArray& deviceStateVec, int numQubits) {
    hipError_t cuda_status;
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
        printf("hipSetDevice failed! ");
        exit(1);
	}
    assert(numQubits > (SINGLE_SIZE_DEP +THREAD_DEP + 1 + REDUCE_BLOCK_DEP + THREAD_DEP + 1));
    assert(numQubits < 31);
    size_t size = sizeof(qreal) << numQubits;
    checkCudaErrors(hipMalloc(&deviceStateVec.real, size));
    checkCudaErrors(hipMalloc(&deviceStateVec.imag, size));
    checkCudaErrors(hipMemset(deviceStateVec.real, 0, size));
    checkCudaErrors(hipMemset(deviceStateVec.imag, 0, size));
    qreal one = 1;
    checkCudaErrors(hipMemcpy(deviceStateVec.real, &one, sizeof(qreal), hipMemcpyHostToDevice)); // state[0] = 1
}


#define SINGLE_GATE_BEGIN \
    qindex idx = blockIdx.x * blockSize + threadIdx.x; \
    qindex mask = (qindex(1) << targetQubit) - 1; \
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        qindex hi = lo | (qindex(1) << targetQubit);

#define SINGLE_GATE_END }

#define CONTROL_GATE_BEGIN \
    qindex idx = blockIdx.x * blockSize + threadIdx.x; \
    qindex mask = (qindex(1) << targetQubit) - 1; \
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) { \
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask); \
        if (!((lo >> controlQubit) & 1)) \
            continue; \
        qindex hi = lo | (qindex(1) << targetQubit);

#define CONTROL_GATE_END }

template <unsigned int blockSize>
__global__ void controlledNotGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        qreal real = a.real[lo];
        qreal imag = a.imag[lo];
        a.real[lo] = a.real[hi];
        a.imag[lo] = a.imag[hi];
        a.real[hi] = real;
        a.imag[hi] = imag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void controlledPauliYGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = hiImag;
        a.imag[lo] = -hiReal;
        a.real[hi] = -loImag;
        a.imag[hi] = loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void controlledRotateXGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, qreal alpha, qreal beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * hiImag;
        a.imag[lo] = alpha * loImag - beta * hiReal;
        a.real[hi] = alpha * hiReal + beta * loImag;
        a.imag[hi] = alpha * hiImag - beta * loReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void controlledRotateYGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, qreal alpha, qreal beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal - beta * hiReal;
        a.imag[lo] = alpha * loImag - beta * hiImag;
        a.real[hi] = beta * loReal + alpha * hiReal;
        a.imag[hi] = beta * loImag + alpha * hiImag;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void controlledRotateZGate(ComplexArray a, int numQubit_, int controlQubit, int targetQubit, qreal alpha, qreal beta) {
    CONTROL_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * loImag;
        a.imag[lo] = alpha * loImag - beta * loReal;
        a.real[hi] = alpha * hiReal - beta * hiImag;
        a.imag[hi] = alpha * hiImag + beta * hiReal;
    } CONTROL_GATE_END
}

template <unsigned int blockSize>
__global__ void hadamardGate(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = recRoot2 * (loReal + hiReal);
        a.imag[lo] = recRoot2 * (loImag + hiImag);
        a.real[hi] = recRoot2 * (loReal - hiReal);
        a.imag[hi] = recRoot2 * (loImag - hiImag);
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void pauliXGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal real = a.real[lo];
        qreal imag = a.imag[lo];
        a.real[lo] = a.real[hi];
        a.imag[lo] = a.imag[hi];
        a.real[hi] = real;
        a.imag[hi] = imag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void pauliYGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = hiImag;
        a.imag[lo] = -hiReal;
        a.real[hi] = -loImag;
        a.imag[hi] = loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void pauliZGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        a.real[hi] = -a.real[hi];
        a.imag[hi] = -a.imag[hi];
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void rotateXGate(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * hiImag;
        a.imag[lo] = alpha * loImag - beta * hiReal;
        a.real[hi] = alpha * hiReal + beta * loImag;
        a.imag[hi] = alpha * hiImag - beta * loReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void rotateYGate(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal - beta * hiReal;
        a.imag[lo] = alpha * loImag - beta * hiImag;
        a.real[hi] = beta * loReal + alpha * hiReal;
        a.imag[hi] = beta * loImag + alpha * hiImag;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void rotateZGate(ComplexArray a, int numQubit_, int targetQubit, qreal alpha, qreal beta) {
    SINGLE_GATE_BEGIN {
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[lo] = alpha * loReal + beta * loImag;
        a.imag[lo] = alpha * loImag - beta * loReal;
        a.real[hi] = alpha * hiReal - beta * hiImag;
        a.imag[hi] = alpha * hiImag + beta * hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void sGate(ComplexArray a, int numQubit_, int targetQubit) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = -hiImag;
        a.imag[hi] = hiReal;
    } SINGLE_GATE_END
}

template <unsigned int blockSize>
__global__ void tGate(ComplexArray a, int numQubit_, int targetQubit, qreal recRoot2) {
    SINGLE_GATE_BEGIN {
        qreal hiReal = a.real[hi];
        qreal hiImag = a.imag[hi];
        a.real[hi] = recRoot2 * (hiReal - hiImag);
        a.imag[hi] = recRoot2 * (hiReal + hiImag);
    } SINGLE_GATE_END
}

void kernelExecSimple(ComplexArray& deviceStateVec, int numQubits, const Schedule& schedule) {
    int numQubit_ = numQubits - 1;
    int nVec = 1 << numQubit_;
    for (auto& gg: schedule.gateGroups) {
        for (auto& gate: gg.gates) {
            switch (gate.type) {
                case GateCNot: {
                    controlledNotGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                    break;
                }
                case GateCPauliY: {
                    controlledPauliYGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit);
                    break;
                }
                case GateCRotateX: {
                    controlledRotateXGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real, gate.mat[0][1].imag);
                    break;
                }
                case GateCRotateY: {
                    controlledRotateYGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real, gate.mat[1][0].real);
                    break;
                }
                case GateCRotateZ: {
                    controlledRotateZGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.controlQubit, gate.targetQubit, gate.mat[0][0].real, - gate.mat[0][0].imag);
                    break;
                }
                case GateHadamard: {
                    hadamardGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                    break;
                }
                case GatePauliX: {
                    pauliXGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GatePauliY: {
                    pauliYGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GatePauliZ: {
                    pauliZGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GateRotateX: {
                    rotateXGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real, gate.mat[0][1].imag);
                    break;
                }
                case GateRotateY: {
                    rotateYGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real, gate.mat[1][0].real);
                    break;
                }
                case GateRotateZ: {
                    rotateZGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(
                        deviceStateVec, numQubit_, gate.targetQubit, gate.mat[0][0].real, - gate.mat[0][0].imag);
                    break;
                }
                case GateS: {
                    sGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit);
                    break;
                }
                case GateT: {
                    tGate<1<<THREAD_DEP><<<nVec>>(SINGLE_SIZE_DEP + THREAD_DEP), 1<<THREAD_DEP>>>(deviceStateVec, numQubit_, gate.targetQubit, 1/sqrt(2));
                    break;
                }
                default: {
                    assert(false);
                }
            }
        }
    }
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__device__ void blockReduce(volatile qreal *sdata, unsigned int tid) {
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
}

template <unsigned int blockSize>
__global__ void reduce(qreal* g_idata, qreal *g_odata, unsigned int n, unsigned int gridSize) {
    __shared__ qreal sdata[blockSize];
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockSize + threadIdx.x;
    unsigned twoGrid = gridSize << 1;
    sdata[tid] = 0;
    for (int i = idx; i < n; i += twoGrid) {
        sdata[tid] += g_idata[i] + g_idata[i + gridSize];
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__global__ void measure(ComplexArray a, qreal* ans, int numQubit_, int targetQubit) {
    __shared__ qreal sdata[blockSize];
    qindex idx = blockIdx.x * blockSize + threadIdx.x;
    int tid = threadIdx.x;
    qindex mask = (qindex(1) << targetQubit) - 1;
    sdata[tid] = 0;
    for (qindex i = (idx << SINGLE_SIZE_DEP); i < ((idx + 1) << SINGLE_SIZE_DEP); i++) {
        qindex lo = ((i >> targetQubit) << (targetQubit + 1)) | (i & mask);
        qreal loReal = a.real[lo];
        qreal loImag = a.imag[lo];
        sdata[tid] += loReal * loReal + loImag * loImag;
    }
    __syncthreads();
    blockReduce<blockSize>(sdata, tid);
    if (tid == 0) ans[blockIdx.x] = sdata[0];
}

qreal kernelMeasure(ComplexArray& deviceStateVec, int numQubits, int targetQubit) {
    int numQubit_ = numQubits - 1;
    qindex nVec = 1 << numQubit_;
    qindex totalBlocks = nVec >> THREAD_DEP >> SINGLE_SIZE_DEP;
    qreal *ans1, *ans2, *ans3;
    checkCudaErrors(hipMalloc(&ans1, sizeof(qreal) * totalBlocks));
    measure<1<<THREAD_DEP><<<totalBlocks, 1<<THREAD_DEP>>>(deviceStateVec, ans1, numQubit_, targetQubit);
    checkCudaErrors(hipMalloc(&ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    reduce<1<<THREAD_DEP><<<1<<REDUCE_BLOCK_DEP, 1<<THREAD_DEP>>>
        (ans1, ans2, totalBlocks, 1 << (THREAD_DEP + REDUCE_BLOCK_DEP));
    checkCudaErrors(hipHostMalloc(&ans3, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP)));
    checkCudaErrors(hipMemcpy(ans3, ans2, sizeof(qreal) * (1<<REDUCE_BLOCK_DEP), hipMemcpyDeviceToHost));
    qreal ret = 0;
    for (int i = 0; i < (1<<REDUCE_BLOCK_DEP); i++)
        ret += ans3[i];
    checkCudaErrors(hipFree(ans1));
    checkCudaErrors(hipFree(ans2));
    checkCudaErrors(hipHostFree(ans3));
    return ret;
}

Complex kernelGetAmp(ComplexArray& deviceStateVec, qindex idx) {
    Complex ret;
    hipMemcpy(&ret.real, deviceStateVec.real + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    hipMemcpy(&ret.imag, deviceStateVec.imag + idx, sizeof(qreal), hipMemcpyDeviceToHost);
    return ret;
}